#include "hip/hip_runtime.h"
#include <arrayfire.h>
#include <af/util.h>
#include <hip/hip_runtime.h>

using namespace af;

array cexp(const array &in)
{
	if (!in.iscomplex()) return exp(in);
	return exp(real(in))*complex(cos(imag(in)), sin(imag(in)));
}

static const double elements_Image[] =
{
	0.1117, 0.5606, 0.6126, 0.8452, 0.8908, 0.4843, 0.4896, 0.5386, 0.2703, 2, 2,
	0.1363, 0.9296, 0.9900, 0.7386, 0.9823, 0.8449, 0.1925, 0.6952, 0.2085, 2, 2,
	0.6787, 0.6967, 0.5277, 0.5860, 0.7690, 0.2094, 0.1231, 0.4991, 0.5650, 2, 2,
	0.4952, 0.5828, 0.4795, 0.2467, 0.5814, 0.5523, 0.2055, 0.5358, 0.6403, 2, 2,
	0.1897, 0.8154, 0.8013, 0.6664, 0.9283, 0.6299, 0.1465, 0.4452, 0.4170, 2, 2,
	0.4950, 0.8790, 0.2278, 0.0835, 0.5801, 0.0320, 0.1891, 0.1239, 0.2060, 2, 2,
	0.1476, 0.9889, 0.4981, 0.6260, 0.0170, 0.6147, 0.0427, 0.4904, 0.9479, 2, 2,
	0.0550, 0.0005, 0.9009, 0.6609, 0.1209, 0.3624, 0.6352, 0.8530, 0.0821, 2, 2,
	0.8507, 0.8654, 0.5747, 0.7298, 0.8627, 0.0495, 0.2819, 0.8739, 0.1057, 2, 2,
	0.5386, 0.2703, 0.1420, 0.8604, 0.0309, 0.5590, 0.1182, 0.8182, 0.9052, 2, 2,
	0.6952, 0.2085, 0.1665, 0.9344, 0.9391, 0.8541, 0.9884, 0.1002, 0.6754, 2, 2
}; // Additional lines for 45 degrees test

void SplitMultiplyComplex(array& A, array&  B, array& realRealPart, array& realImagPart, array& imagRealPart, array& imagImagPart)
{
	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	//array A, B;
	//array realRealPart;             // ac
	//array realImagPart;				// ad
	//array imagRealPart;				// bc
	//array imagImagPart;				// bd
	realRealPart = real(A)*real(B);
	realImagPart = real(A)*imag(B);
	imagRealPart = imag(A)*real(B);
	imagImagPart = imag(A)*imag(B);
}

array Compute2DColumnwise_FrFTUniform(array & Image2D, array& ColumnScales_1D, int d_NoOfElements, int d_NoOfScales)
{
	int debugFlag_ToPrint = 0;
	
	/*-----------------------------------Preparing Padded & Tiled Imag2D --------------------------------------------*/
	array Zeros = constant(0, d_NoOfElements, d_NoOfElements, f32);
	array Zero_Padded_Image2D = join(0, Image2D, Zeros);
	//af_print(Zero_Padded_Image2D);
	array Image2D_Tiled = tile(Zero_Padded_Image2D, 1, 1, d_NoOfScales);
	//af_print(Image2D_Tiled);

	//printf("Successfully prepared the padded and tiled images of size %d-by-%d-by-%d  !", 2 * d_NoOfElements, d_NoOfElements, d_NoOfScales);
	//printf("\n");

	/*-------------------------------------------Creating Index Cubes and Sequences----------------------------------------------------*/
	int N = d_NoOfElements - 1;
	array leftSideIndexes = array(seq(0, N, 1));
	array rightSideIndexesOnes = -1 * array(seq(1, d_NoOfElements, 1));
	array rightSideIndexesZeros = constant(0, d_NoOfElements, 1);
	array rightSideIndexesN_2 = constant(N / 2, d_NoOfElements, 1);

	if (debugFlag_ToPrint)
	{
		af_print(leftSideIndexes);
		//af_print(leftSideIndexes.isdouble());
		//af_print(flip( rightSideIndexesOnes,0));
		//af_print(rightSideIndexesZeros);
		//af_print(rightSideIndexesN_2);
	}


	array indexedElementsEn = join(0, leftSideIndexes, flip(rightSideIndexesOnes, 0));
	array indexedElementsPre = join(0, leftSideIndexes, rightSideIndexesN_2);     /* This is for Keeping pre and post multiplication factor upper half only*/
	array indexedElementsPost = join(0, leftSideIndexes, rightSideIndexesZeros);
	if (debugFlag_ToPrint)
	{
		//af_print(indexedElementsEn);
		//af_print(indexedElementsPre_Post);
	}


	array indexedElements_Tiled_En = tile(pow(indexedElementsEn, 2), 1, d_NoOfElements, d_NoOfScales);
	array indexedElements_Tiled_PreMulti = tile(indexedElementsPre - N / 2, 1, d_NoOfElements, d_NoOfScales);
	array indexedElements_Tiled_PostMulti = tile(indexedElementsPost, 1, d_NoOfElements, d_NoOfScales);
	
	if (debugFlag_ToPrint)
	{
		af_print(indexedElements_Tiled_En.col(0));
		af_print(indexedElements_Tiled_PreMulti.col(0));
		af_print(indexedElements_Tiled_PostMulti.col(0));
	}


	/*--------------------------Creating FrFT scale cubes------------------------------------*/
	array ColumnScales_1D_FullTiled;
	if (d_NoOfScales == 1)
	{
		//af_print(ColumnScales_1D);
		ColumnScales_1D_FullTiled = constant(ColumnScales_1D.scalar<float>(), 2 * d_NoOfElements, d_NoOfElements);
		//af_print(ColumnScales_1D_FullTiled);
	}
	else
	{
		array ColumnScales_1D_Mods  = moddims(ColumnScales_1D, 1, 1, d_NoOfScales);
		array ColumnScales_1D_Tiled_depth = tile(ColumnScales_1D_Mods, 2 * d_NoOfElements, d_NoOfElements, 1);
		ColumnScales_1D_FullTiled = moddims(ColumnScales_1D_Tiled_depth, 2 * d_NoOfElements, d_NoOfElements, d_NoOfScales);
	}
	
	if (debugFlag_ToPrint)
	{
		//af_print(ColumnScales_1D_Mods);
		//af_print(ColumnScales_1D_Tiled_depth);
		//af_print(ColumnScales_1D_FullTiled);
	}


	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	/*array imaginaryUnit_Tiled = tile(i, 2 * d_NoOfElements, d_NoOfElements, d_NoOfScales);
	if (debugFlag_ToPrint)
	{
		af_print(imaginaryUnit_Tiled.slice(0))
	}*/
	array Seq_En = cexp(complex(0, -af::Pi * indexedElements_Tiled_En * ColumnScales_1D_FullTiled / d_NoOfElements));   /* E(n) as defined in the paper*/

	//array temp = array(seq(0, 5)).as(f64);
	//af_print(temp);
	//af_print(complex(0,temp));
	/*af_print(cexp(-complex(0,af::Pi * indexedElements_Tiled_En * ColumnScales_1D_FullTiled / d_NoOfElements)));
	array Seq_En = cexp(-imaginaryUnit_Tiled*af::Pi * indexedElements_Tiled_En * ColumnScales_1D_FullTiled / d_NoOfElements);
	af_print(Seq_En);*/


	array Ones = constant(1, d_NoOfElements, d_NoOfElements, f32);
	array subtractValues = tile(join(0, Zeros, Ones), 1, 1, d_NoOfScales);			/* This is for  Keeping pre and post multiplication factor upper half only*/
	array PreMultiplicationFactor = cexp(complex (0,  af::Pi * indexedElements_Tiled_PreMulti * ColumnScales_1D_FullTiled * N / d_NoOfElements)) - subtractValues;
	array PostMultiplicationFactor = cexp(complex(0, af::Pi *  indexedElements_Tiled_PostMulti * ColumnScales_1D_FullTiled * N / d_NoOfElements)) - subtractValues;

	if (debugFlag_ToPrint)
	{
		af_print(Seq_En);
		af_print(PreMultiplicationFactor.col(0));
		af_print(PostMultiplicationFactor.col(0));
	}


	/*--------------------Preprocessing Cubes-----------------------*/
	array Image2D_Tiled_PreMulti = Image2D_Tiled * PreMultiplicationFactor;
	array Image2D_Tiled_PreMulti_SeqEn = Image2D_Tiled_PreMulti * Seq_En;

	if (debugFlag_ToPrint)
	{
		af_print(Image2D_Tiled_PreMulti);
		af_print(Seq_En);
		af_print(Image2D_Tiled_PreMulti_SeqEn);
	}


	/*-------------------Computing Convolution--------------------*/
	array firstFFT_X = fft(Image2D_Tiled_PreMulti_SeqEn);
	array secondFFT_X = fft(conjg(Seq_En));
	array interim_FrFT_X = ifft(firstFFT_X * secondFFT_X);

	if (debugFlag_ToPrint)
	{
		af_print(firstFFT_X.col(0));
		af_print(secondFFT_X.col(0));
		af_print(interim_FrFT_X.cols(0, N));
	}



	/*-------------------Postprocessing-----------------------------*/
	array  FrFT_Image_X = interim_FrFT_X * Seq_En * PostMultiplicationFactor;

	/*--------------------Grab only the top half elements drop overlapping------------------*/
	array FrFT_Image_X_Cube = FrFT_Image_X.rows(0, N);

	if (debugFlag_ToPrint)
	{
		af_print(FrFT_Image_X);
		af_print(FrFT_Image_X_Cube.cols(0, N));
	}

	return FrFT_Image_X_Cube;
}

void Compute2DComplementaryLines_FrFTVariableScales(array & OneD_FrFT, array& ColumnScales_1D, array& final2DFrFTImage, array& final2DFrFTConjImage, int d_NoOfElements, int d_NoOfScales)
{
	int debugFlag_ToPrint = 0;

	int N = d_NoOfElements - 1;
	array lineSpacing = array(seq(-N / 2, N / 2));
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();

	array lineSpacing_Square_TiledLevel = tile(lineSpacing_Square, 1, 1, d_NoOfScales);
	//printf("\nCreated preliminary indexed matrices and sequences, E_n, PreMultiplication, PostMultiplication and Linespacing .\n");

	af_print(lineSpacing_Square);
	if (debugFlag_ToPrint)
	{
		//af_print(lineSpacing);
		//af_print(lineSpacing_tiled_Y);
		af_print(lineSpacing_Square);
		af_print(lineSpacing_Square_TiledLevel);
	}

	array beta_Tiled;
	if (d_NoOfScales == 1)
	{
		beta_Tiled = constant(ColumnScales_1D.scalar<float>(), d_NoOfElements, d_NoOfElements);
	}
	else
	{ 
		array beta_Mods = moddims(ColumnScales_1D, 1, 1, d_NoOfScales);
		array beta_Tiled_depth = tile(beta_Mods, d_NoOfElements, d_NoOfElements, 1);
		beta_Tiled = moddims(beta_Tiled_depth, d_NoOfElements, d_NoOfElements, d_NoOfScales);
	}
	//printf("\nCreated alpha and beta matrices.\n");

	if (debugFlag_ToPrint)
	{
		//af_print(beta_Tiled_depth);
		af_print(beta_Tiled);
	}


	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	array BetaFactor = cexp( complex(0, -2 * af::Pi * lineSpacing_Square_TiledLevel * beta_Tiled / d_NoOfElements));


	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	array realRealPart;             // ac
	array realImagPart;				// ad 
	array imagRealPart;				// bc
	array imagImagPart;				// bd

	SplitMultiplyComplex(OneD_FrFT, BetaFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);

	array tempSeq_X = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); // sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor);
	
	array tempSeqConj_X = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart));//  sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor));
	
	if (debugFlag_ToPrint)
	{
		af_print(BetaFactor.cols(0, N));

		af_print(tempSeq_X);
		//af_print(sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor));
		af_print(tempSeqConj_X);
		//af_print(sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor)));
	}

	final2DFrFTImage = moddims(tempSeq_X, d_NoOfElements, d_NoOfScales).T();
	final2DFrFTConjImage = moddims(tempSeqConj_X, d_NoOfElements, d_NoOfScales).T();
}

void FullyVectorized2DPolarTransform(array& Image, int d_NoOfElements, int d_NoOfAngles, int d_NoOfLevels, array& FinalPolarGrid)
{
	int debugFlag_ToPrint = 0;
	int N = d_NoOfElements - 1; /* it is always even as described in the paper*/

	//array Image = randu(d_NoOfElements, d_NoOfElements, f64);
	printf("\nCreating a %d-by-%d elements of an ArrayFire object, it has %d levels.", d_NoOfElements, d_NoOfElements, d_NoOfLevels);
	printf("\n");

	if (debugFlag_ToPrint)
	{
		af_print(Image);
		af_print(Image.row(0));
		af_print(Image.col(0));
	}

	/*--------------------------Creating Alpha Cubes------------------------------------*/
	array alpha_Levels = cos(af::Pi / d_NoOfAngles * array(seq(1, d_NoOfLevels)));
	if (debugFlag_ToPrint)
	{
		af_print(alpha_Levels);
	}
	/*--------------------------Creating Beta Cubes---------------------------------------*/
	array beta_Levels = sin(Pi / d_NoOfAngles * array(seq(1, d_NoOfLevels)));
	//af_print(beta_Levels);
	//printf("\nCreated alpha and beta matrices.\n");

	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	array lineSpacing = array(seq(-N / 2, N / 2));
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
	array ZeroNinty_Factor = cexp(complex(0, -2 * af::Pi * lineSpacing_Square * 1 / d_NoOfElements));
	if (debugFlag_ToPrint)
	{
		//af_print(lineSpacing);
		//af_print(lineSpacing_tiled_Y);
		af_print(lineSpacing_Square);
		af_print(ZeroNinty_Factor);
	}



	/*-------------------- First dimension uniform FrFT for each Image per level-----------------------*/
	array FrFT_Image_X_Cube = Compute2DColumnwise_FrFTUniform(Image.T(), alpha_Levels, d_NoOfElements, d_NoOfLevels);
	array FrFT_Image_Y_Cube = Compute2DColumnwise_FrFTUniform(Image, alpha_Levels, d_NoOfElements, d_NoOfLevels);

	FrFT_Image_X_Cube = FrFT_Image_X_Cube.T();       // Now it needs operation to the other dimension
	FrFT_Image_Y_Cube = FrFT_Image_Y_Cube.T();


	/*--------------------Finally all computations for  the  Polar Grid-----------*/
	//   Computing for all the grid expect two special indexes
	array levelSeq = array(seq(0, d_NoOfLevels - 1));

	array finalIndexSeq1_X = 1 + levelSeq;

	array finalIndexSeq2_X = d_NoOfAngles - finalIndexSeq1_X;

	array finalIndexSeq3_Y = d_NoOfAngles / 2 - finalIndexSeq1_X;

	array finalIndexSeq4_Y = d_NoOfAngles / 2 + finalIndexSeq1_X;

	if (debugFlag_ToPrint)
	{
		af_print(levelSeq);
		af_print(finalIndexSeq1_X);
		af_print(finalIndexSeq2_X);
		af_print(finalIndexSeq3_Y);
		af_print(finalIndexSeq4_Y);
	}

	//array A = constant(1, 10);
	//array B = A; // B and A point to same memory
	//float *d_A = A.device<float>();
	//hipMemset(d_A, 0, A.bytes());
	//af_print(B); //  all zeros since pointed to same memory
	//array element = finalIndexSeq3_Y.row(0);
	//af_print(element);
	//float rho = element.scalar<float>();
	//printf("%f", rho);

	array finalSeq_X, finalSeqConj_X;
	Compute2DComplementaryLines_FrFTVariableScales(FrFT_Image_X_Cube, beta_Levels, finalSeq_X, finalSeqConj_X, d_NoOfElements, d_NoOfLevels);
	finalSeqConj_X = flip(finalSeqConj_X, 1);             // Special operation
	if (debugFlag_ToPrint)
	{
		af_print(finalSeq_X);
		af_print(finalSeqConj_X);
	}

	array finalSeq_Y, finalSeqConj_Y;
	Compute2DComplementaryLines_FrFTVariableScales(FrFT_Image_Y_Cube, beta_Levels, finalSeq_Y, finalSeqConj_Y, d_NoOfElements, d_NoOfLevels);


	// Removing just 2 redundant computations for 45 degree incase
	if (0 == remainder(d_NoOfAngles, 4))
	{
		finalIndexSeq3_Y = finalIndexSeq3_Y.rows(0, d_NoOfLevels - 2);          // Removing just the last rows from 4 structures
		finalSeq_Y = finalSeq_Y.rows(0, d_NoOfLevels - 2);
		finalIndexSeq4_Y = finalIndexSeq4_Y.rows(0, d_NoOfLevels - 2);
		finalSeqConj_Y = finalSeqConj_Y.rows(0, d_NoOfLevels - 2);
	}

	if (debugFlag_ToPrint)
	{
		af_print(finalIndexSeq3_Y);
		af_print(finalIndexSeq4_Y);
		af_print(finalSeq_Y);
		af_print(finalSeqConj_Y);
	}


	//   Computing seperately for two special indexes
	float zeroIndex = 0;
	float nintyIndex = d_NoOfAngles / 2;
	float values[] = { zeroIndex, nintyIndex };
	array SpecialTwoIndexes(2, 1, values);


	array ZeroLineFrFT_Image_X_Cube = FrFT_Image_Y_Cube.slice(zeroIndex).col(N / 2);

	array NintyLineFrFT_Image_Y_Cube = FrFT_Image_X_Cube.slice(zeroIndex).col(N / 2);


	if (debugFlag_ToPrint)
	{
		af_print(FrFT_Image_Y_Cube);
		af_print(ZeroLineFrFT_Image_X_Cube);
		//af_print(FrFT_Image_X_Cube);
		af_print(NintyLineFrFT_Image_Y_Cube);
	}


	array DFTZeroLine = sum(tile(ZeroLineFrFT_Image_X_Cube, 1, d_NoOfElements) *ZeroNinty_Factor);

	array DFTNinetyLine = sum(tile(NintyLineFrFT_Image_Y_Cube, 1, d_NoOfElements) *ZeroNinty_Factor);

	array SpecialTwoLines = join(0, DFTZeroLine, DFTNinetyLine);

	if (debugFlag_ToPrint)
	{
		af_print(DFTZeroLine);
		af_print(DFTNinetyLine);
		af_print(SpecialTwoLines);
	}

	array UnsortedIndexes = join(0, join(0, join(0, join(0, finalIndexSeq1_X, finalIndexSeq2_X), finalIndexSeq3_Y), finalIndexSeq4_Y), SpecialTwoIndexes);
	array tiledUnsortedIndexes = tile(UnsortedIndexes, 1, d_NoOfElements);
	array UnsortedPolarGrid = join(0, join(0, join(0, join(0, finalSeq_X, finalSeqConj_X), finalSeq_Y), finalSeqConj_Y), SpecialTwoLines);

	if (debugFlag_ToPrint)
	{
		af_print(tiledUnsortedIndexes);
		af_print(UnsortedPolarGrid);
	}

	array FinalPolarGridReal;// = constant(0, d_NoOfElements, d_NoOfAngles, c64);
	array Output_Keys_Sorted;
	sort(Output_Keys_Sorted, FinalPolarGridReal, tiledUnsortedIndexes, real(UnsortedPolarGrid));
	if (debugFlag_ToPrint)
	{
		af_print(Output_Keys_Sorted);
		af_print(FinalPolarGridReal);
	}

	array FinalPolarGridImag;// = constant(0, d_NoOfElements, d_NoOfAngles, c64);
	array Output_Keys_Sorted2;
	sort(Output_Keys_Sorted2, FinalPolarGridImag, tiledUnsortedIndexes, imag(UnsortedPolarGrid));
	
	// Put it in the output
	FinalPolarGrid = complex(FinalPolarGridReal, FinalPolarGridImag);
	if (debugFlag_ToPrint)
	{
		af_print(FinalPolarGrid);
	}
	
}

void VectorizedPerLevel2DPolarTransform(array& Image, int d_NoOfElements, int d_NoOfAngles, int d_NoOfLevels, array& FinalPolarGrid)
{

	int debugFlag_ToPrint = 0;
	int N = d_NoOfElements - 1; /* it is always even as described in the paper*/

	array FrFT_Image_X_Cube, FrFT_Image_Y_Cube;
	array finalSeq_X, finalSeqConj_X;
	array finalSeq_Y, finalSeqConj_Y;
	for (int l = 1; l <= d_NoOfLevels; ++l)
	{
		double angle = l * Pi / d_NoOfAngles;
		array alpha = constant( cos(af::Pi / d_NoOfAngles * l),1,1 );
		array beta  = constant ( sin(af::Pi / d_NoOfAngles * l),1,1);

		//  X-axis Level
		/*-------------------- First dimension uniform FrFT for each Image per level-----------------------*/
		FrFT_Image_X_Cube = Compute2DColumnwise_FrFTUniform(Image.T(), alpha, d_NoOfElements, 1);
		FrFT_Image_X_Cube = FrFT_Image_X_Cube.T();       // Now it needs operation to the other dimension
		//af_print(FrFT_Image_X_Cube);
		
		Compute2DComplementaryLines_FrFTVariableScales((FrFT_Image_X_Cube), beta, finalSeq_X, finalSeqConj_X, d_NoOfElements, 1);
		finalSeqConj_X = flip(finalSeqConj_X, 1);             // Special operation

		/*af_print(finalSeq_X);
		af_print(finalSeqConj_X);
		af_print(FinalPolarGrid(l, span));
		af_print(FinalPolarGrid(d_NoOfAngles - l-1, span));*/
		FinalPolarGrid(l, span) = finalSeq_X;
		FinalPolarGrid(d_NoOfAngles - l, span) = finalSeqConj_X;

		if (l * 180 / d_NoOfAngles == 45)
			continue;
		
		// Y-axis level
		/*-------------------- First dimension uniform FrFT for each Image per level-----------------------*/
		FrFT_Image_Y_Cube = Compute2DColumnwise_FrFTUniform((Image), alpha, d_NoOfElements, 1);
		FrFT_Image_Y_Cube = FrFT_Image_Y_Cube.T();

		Compute2DComplementaryLines_FrFTVariableScales(FrFT_Image_Y_Cube, beta, finalSeq_Y, finalSeqConj_Y, d_NoOfElements, 1);

		FinalPolarGrid(d_NoOfAngles / 2 - l, span) = finalSeq_Y;
		FinalPolarGrid(d_NoOfAngles / 2 + l, span) = finalSeqConj_Y;
	}

	//   Computing seperately for two special indexes
	float zeroIndex = 0;
	float nintyIndex = d_NoOfAngles / 2;
	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	array lineSpacing = array(seq(-N / 2, N / 2));
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
	array ZeroNinty_Factor = cexp(complex(0, -2 * af::Pi * lineSpacing_Square * 1 / d_NoOfElements));
	if (debugFlag_ToPrint)
	{
		//af_print(lineSpacing);
		//af_print(lineSpacing_tiled_Y);
		af_print(lineSpacing_Square);
		af_print(ZeroNinty_Factor);
	}

	array ZeroLineFrFT_Image_X_Cube = FrFT_Image_Y_Cube.slice(zeroIndex).col(N / 2);
	array NintyLineFrFT_Image_Y_Cube = FrFT_Image_X_Cube.slice(zeroIndex).col(N / 2);
	if (debugFlag_ToPrint)
	{
		af_print(ZeroLineFrFT_Image_X_Cube);
		af_print(NintyLineFrFT_Image_Y_Cube);
	}

	array DFTZeroLine = sum(tile(ZeroLineFrFT_Image_X_Cube, 1, d_NoOfElements) *ZeroNinty_Factor);
	array DFTNinetyLine = sum(tile((NintyLineFrFT_Image_Y_Cube), 1, d_NoOfElements) *ZeroNinty_Factor);

	FinalPolarGrid(zeroIndex, span) = DFTZeroLine;
	FinalPolarGrid(nintyIndex, span) = DFTNinetyLine;

}

int main(int argc, char* argv[])
{
	try {
		info();
	
		//int myN = 50;
		//array testSize = randn(2 * myN, myN, c64);
		//array tiledTest = tile(testSize, 1, 1, myN / 4); 
		//array FFT_Test = fft(tiledTest);          // MAx capacity is 400
		//

		//array SimpleSeq = array(seq(1, 15));
		////af_print(SimpleSeq);
		//array FFT1DSimple = fft(SimpleSeq);
		////af_print(FFT1DSimple);
		//array IFFTSimple = ifft(FFT1DSimple);
		////af_print(IFFTSimple / pow(15, 2));

		int d_NoOfElements =  11; // 245;// 11; //9         // 2501  is the max tested
		int d_NoOfAngles = d_NoOfElements + 1;

		int d_NoOfLevels;
		if (((d_NoOfAngles - 2) % 4) != 0)
		{
			d_NoOfLevels = (d_NoOfAngles - 2) / 4 +1;
		}
		else
		{
			d_NoOfLevels = (d_NoOfAngles - 2) / 4 ;
		}
		

		array Image(d_NoOfElements, d_NoOfElements, elements_Image);

		//array Image = randu(d_NoOfElements, d_NoOfElements, f32);

		Image = Image.as(f32);
		Image = Image.T();
		array FinalPolarGrid;
		
		FullyVectorized2DPolarTransform( Image,  d_NoOfElements, d_NoOfAngles, d_NoOfLevels, FinalPolarGrid);
		printf("\nCompleted Fully Vectorized version");
		af_print(FinalPolarGrid);

		array NonVectorizedPolarGrid = constant(0, d_NoOfAngles, d_NoOfElements, c32);       // Pre allocated here
		VectorizedPerLevel2DPolarTransform(Image, d_NoOfElements, d_NoOfAngles, d_NoOfLevels, NonVectorizedPolarGrid);
		printf("\nCompleted Fully Vectorized-per-level version\n");
		af_print(NonVectorizedPolarGrid);

		array MaxAbsError = (sum(abs(FinalPolarGrid) - abs(NonVectorizedPolarGrid)));
		printf("\nMaximum Absolute error between fully vectorized and vectorized per level");
		af_print(sum(MaxAbsError));

		printf("Successfully completed the computations of 2D DFT on a full Polar Grid !\n");
		

	}
	catch (af::exception& e) {
		fprintf(stderr, "%s\n", e.what());
		throw;
	}

#ifdef WIN32 // pause in Windows
	if (!(argc == 2 && argv[1][0] == '-')) {
		printf("hit [enter]...");
		getchar();
	}
#endif
	return 0;
}
