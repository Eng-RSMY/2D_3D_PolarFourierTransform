#include "hip/hip_runtime.h"
/*
* This is a program for testing the 3D Spherical Transform
*/

#include <arrayfire.h>
#include <af/util.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <windows.h>
#include <tlhelp32.h>
#include <stdio.h>

using namespace af;


enum AXES
{
	X_AXIS = 1,
	Y_AXIS = 0,   // This is how arrayFire refers to Y-axis
	Z_AXIS = 2
};

static const af::dtype PRECISION_REAL = f64;                   // Change here from single to double
static const af::dtype PRECISION_COMPLEX = c64;


array cexp(const array &in)
{
	if (!in.iscomplex()) return exp(in);
	return exp(real(in))*complex(cos(imag(in)), sin(imag(in)));
}

// This is a test image constant so we can compare exact values of the lines data computed

static const double ThreeDImageValues[] = 
{
	0.8147, 0.9649, 0.7922, 0.3922, 0.6948, 0.4898, 0.1190, 0.6991, 0.8143,
	0.9058, 0.1576, 0.9595, 0.6555, 0.3171, 0.4456, 0.4984, 0.8909, 0.2435,
	0.1270, 0.9706, 0.6557, 0.1712, 0.9502, 0.6463, 0.9597, 0.9593, 0.9293,
	0.9134, 0.9572, 0.0357, 0.7060, 0.0344, 0.7094, 0.3404, 0.5472, 0.3500,
	0.6324, 0.4854, 0.8491, 0.0318, 0.4387, 0.7547, 0.5853, 0.1386, 0.1966,
	0.0975, 0.8003, 0.9340, 0.2769, 0.3816, 0.2760, 0.2238, 0.1493, 0.2511,
	0.2785, 0.1419, 0.6787, 0.0462, 0.7655, 0.6797, 0.7513, 0.2575, 0.6160,
	0.5469, 0.4218, 0.7577, 0.0971, 0.7952, 0.6551, 0.2551, 0.8407, 0.4733,
	0.9575, 0.9157, 0.7431, 0.8235, 0.1869, 0.1626, 0.5060, 0.2543, 0.3517, 

	//I(:, : , 2) =

	0.8308, 0.0759, 0.3371, 0.6892, 0.9961, 0.0844, 0.1361, 0.4018, 0.9448,
	0.5853, 0.0540, 0.1622, 0.7482, 0.0782, 0.3998, 0.8693, 0.0760, 0.4909,
	0.5497, 0.5308, 0.7943, 0.4505, 0.4427, 0.2599, 0.5797, 0.2399, 0.4893,
	0.9172, 0.7792, 0.3112, 0.0838, 0.1067, 0.8001, 0.5499, 0.1233, 0.3377,
	0.2858, 0.9340, 0.5285, 0.2290, 0.9619, 0.4314, 0.1450, 0.1839, 0.9001,
	0.7572, 0.1299, 0.1656, 0.9133, 0.0046, 0.9106, 0.8530, 0.2400, 0.3692,
	0.7537, 0.5688, 0.6020, 0.1524, 0.7749, 0.1818, 0.6221, 0.4173, 0.1112,
	0.3804, 0.4694, 0.2630, 0.8258, 0.8173, 0.2638, 0.3510, 0.0497, 0.7803,
	0.5678, 0.0119, 0.6541, 0.5383, 0.8687, 0.1455, 0.5132, 0.9027, 0.3897,

	//I(:, : , 3) =

	0.2417, 0.3532, 0.5470, 0.0811, 0.8176, 0.5502, 0.2259, 0.9797, 0.2217,
	0.4039, 0.8212, 0.2963, 0.9294, 0.7948, 0.6225, 0.1707, 0.4389, 0.1174,
	0.0965, 0.0154, 0.7447, 0.7757, 0.6443, 0.5870, 0.2277, 0.1111, 0.2967,
	0.1320, 0.0430, 0.1890, 0.4868, 0.3786, 0.2077, 0.4357, 0.2581, 0.3188,
	0.9421, 0.1690, 0.6868, 0.4359, 0.8116, 0.3012, 0.3111, 0.4087, 0.4242,
	0.9561, 0.6491, 0.1835, 0.4468, 0.5328, 0.4709, 0.9234, 0.5949, 0.5079,
	0.5752, 0.7317, 0.3685, 0.3063, 0.3507, 0.2305, 0.4302, 0.2622, 0.0855,
	0.0598, 0.6477, 0.6256, 0.5085, 0.9390, 0.8443, 0.1848, 0.6028, 0.2625,
	0.2348, 0.4509, 0.7802, 0.5108, 0.8759, 0.1948, 0.9049, 0.7112, 0.8010,

	//I(:, : , 4) =

	0.0292, 0.5211, 0.8852, 0.1068, 0.1978, 0.8055, 0.7127, 0.8181, 0.4538,
	0.9289, 0.2316, 0.9133, 0.6538, 0.0305, 0.5767, 0.5005, 0.8175, 0.4324,
	0.7303, 0.4889, 0.7962, 0.4942, 0.7441, 0.1829, 0.4711, 0.7224, 0.8253,
	0.4886, 0.6241, 0.0987, 0.7791, 0.5000, 0.2399, 0.0596, 0.1499, 0.0835,
	0.5785, 0.6791, 0.2619, 0.7150, 0.4799, 0.8865, 0.6820, 0.6596, 0.1332,
	0.2373, 0.3955, 0.3354, 0.9037, 0.9047, 0.0287, 0.0424, 0.5186, 0.1734,
	0.4588, 0.3674, 0.6797, 0.8909, 0.6099, 0.4899, 0.0714, 0.9730, 0.3909,
	0.9631, 0.9880, 0.1366, 0.3342, 0.6177, 0.1679, 0.5216, 0.6490, 0.8314,
	0.5468, 0.0377, 0.7212, 0.6987, 0.8594, 0.9787, 0.0967, 0.8003, 0.8034,

	//I(:, : , 5) =

	0.0605, 0.9841, 0.0527, 0.7011, 0.0326, 0.8555, 0.3846, 0.3439, 0.4253,
	0.3993, 0.1672, 0.7379, 0.6663, 0.5612, 0.6448, 0.5830, 0.5841, 0.3127,
	0.5269, 0.1062, 0.2691, 0.5391, 0.8819, 0.3763, 0.2518, 0.1078, 0.1615,
	0.4168, 0.3724, 0.4228, 0.6981, 0.6692, 0.1909, 0.2904, 0.9063, 0.1788,
	0.6569, 0.1981, 0.5479, 0.6665, 0.1904, 0.4283, 0.6171, 0.8797, 0.4229,
	0.6280, 0.4897, 0.9427, 0.1781, 0.3689, 0.4820, 0.2653, 0.8178, 0.0942,
	0.2920, 0.3395, 0.4177, 0.1280, 0.4607, 0.1206, 0.8244, 0.2607, 0.5985,
	0.4317, 0.9516, 0.9831, 0.9991, 0.9816, 0.5895, 0.9827, 0.5944, 0.4709,
	0.0155, 0.9203, 0.3015, 0.1711, 0.1564, 0.2262, 0.7302, 0.0225, 0.6959,

	//I(:, : , 6) =

	0.6999, 0.7184, 0.2665, 0.6377, 0.2240, 0.9160, 0.0358, 0.2428, 0.5466,
	0.6385, 0.9686, 0.1537, 0.9577, 0.6678, 0.0012, 0.1759, 0.9174, 0.4257,
	0.0336, 0.5313, 0.2810, 0.2407, 0.8444, 0.4624, 0.7218, 0.2691, 0.6444,
	0.0688, 0.3251, 0.4401, 0.6761, 0.3445, 0.4243, 0.4735, 0.7655, 0.6476,
	0.3196, 0.1056, 0.5271, 0.2891, 0.7805, 0.4609, 0.1527, 0.1887, 0.6790,
	0.5309, 0.6110, 0.4574, 0.6718, 0.6753, 0.7702, 0.3411, 0.2875, 0.6358,
	0.6544, 0.7788, 0.8754, 0.6951, 0.0067, 0.3225, 0.6074, 0.0911, 0.9452,
	0.4076, 0.4235, 0.5181, 0.0680, 0.6022, 0.7847, 0.1917, 0.5762, 0.2089,
	0.8200, 0.0908, 0.9436, 0.2548, 0.3868, 0.4714, 0.7384, 0.6834, 0.7093,

	//I(:, : , 7) =

	0.2362, 0.4162, 0.3181, 0.7210, 0.3658, 0.0938, 0.3477, 0.3592, 0.2703,
	0.1194, 0.8419, 0.1192, 0.5225, 0.7635, 0.5254, 0.1500, 0.7363, 0.1971,
	0.6073, 0.8329, 0.9398, 0.9937, 0.6279, 0.5303, 0.5861, 0.3947, 0.8217,
	0.4501, 0.2564, 0.6456, 0.2187, 0.7720, 0.8611, 0.2621, 0.6834, 0.4299,
	0.4587, 0.6135, 0.4795, 0.1058, 0.9329, 0.4849, 0.0445, 0.7040, 0.8878,
	0.6619, 0.5822, 0.6393, 0.1097, 0.9727, 0.3935, 0.7549, 0.4423, 0.3912,
	0.7703, 0.5407, 0.5447, 0.0636, 0.1920, 0.6714, 0.2428, 0.0196, 0.7691,
	0.3502, 0.8699, 0.6473, 0.4046, 0.1389, 0.7413, 0.4424, 0.3309, 0.3968,
	0.6620, 0.2648, 0.5439, 0.4484, 0.6963, 0.5201, 0.6878, 0.4243, 0.8085,

	//I(:, : , 8) =

	0.7551, 0.7689, 0.4070, 0.6787, 0.6967, 0.5277, 0.5860, 0.7690, 0.2094,
	0.3774, 0.1673, 0.7487, 0.4952, 0.5828, 0.4795, 0.2467, 0.5814, 0.5523,
	0.2160, 0.8620, 0.8256, 0.1897, 0.8154, 0.8013, 0.6664, 0.9283, 0.6299,
	0.7904, 0.9899, 0.7900, 0.4950, 0.8790, 0.2278, 0.0835, 0.5801, 0.0320,
	0.9493, 0.5144, 0.3185, 0.1476, 0.9889, 0.4981, 0.6260, 0.0170, 0.6147,
	0.3276, 0.8843, 0.5341, 0.0550, 0.0005, 0.9009, 0.6609, 0.1209, 0.3624,
	0.6713, 0.5880, 0.0900, 0.8507, 0.8654, 0.5747, 0.7298, 0.8627, 0.0495,
	0.4386, 0.1548, 0.1117, 0.5606, 0.6126, 0.8452, 0.8908, 0.4843, 0.4896,
	0.8335, 0.1999, 0.1363, 0.9296, 0.9900, 0.7386, 0.9823, 0.8449, 0.1925,

	//I(:, : , 9) =

	0.1231, 0.4991, 0.5650, 0.6210, 0.9844, 0.3013, 0.3479, 0.5400, 0.1781,
	0.2055, 0.5358, 0.6403, 0.5737, 0.8589, 0.2955, 0.4460, 0.7069, 0.3596,
	0.1465, 0.4452, 0.4170, 0.0521, 0.7856, 0.3329, 0.0542, 0.9995, 0.0567,
	0.1891, 0.1239, 0.2060, 0.9312, 0.5134, 0.4671, 0.1771, 0.2878, 0.5219,
	0.0427, 0.4904, 0.9479, 0.7287, 0.1776, 0.6482, 0.6628, 0.4145, 0.3358,
	0.6352, 0.8530, 0.0821, 0.7378, 0.3986, 0.0252, 0.3308, 0.4648, 0.1757,
	0.2819, 0.8739, 0.1057, 0.0634, 0.1339, 0.8422, 0.8985, 0.7640, 0.2089,
	0.5386, 0.2703, 0.1420, 0.8604, 0.0309, 0.5590, 0.1182, 0.8182, 0.9052,
	0.6952, 0.2085, 0.1665, 0.9344, 0.9391, 0.8541, 0.9884, 0.1002, 0.6754
};




void SplitMultiplyComplex(array& A, array&  B, array& realRealPart, array& realImagPart, array& imagRealPart, array& imagImagPart)
{
	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	//array A, B;
	//array realRealPart;             // ac
	//array realImagPart;				// ad
	//array imagRealPart;				// bc
	//array imagImagPart;				// bd
	realRealPart = real(A)*real(B);
	//af_print(realRealPart);
	realImagPart = real(A)*imag(B);
	imagRealPart = imag(A)*real(B);
	imagImagPart = imag(A)*imag(B);
}


static array indexedElements_Tiled_En;
static array indexedElements_Tiled_PreMulti;
static array indexedElements_Tiled_PostMulti;

void GlobalArraysComputeInitialize(int d_NoOfElements)
{
	/*-------------------------------------------Creating Index Cubes and Sequences----------------------------------------------------*/
	int N = d_NoOfElements - 1;
	array leftSideIndexes = array(seq(0, N, 1)).as(PRECISION_REAL);
	// af_print(leftSideIndexes);
	array rightSideIndexesOnes = -1 * array(seq(1, d_NoOfElements, 1)).as(PRECISION_REAL);
	//af_print(flip( rightSideIndexesOnes,0));
	array rightSideIndexesZeros = constant(0, d_NoOfElements, 1, PRECISION_REAL);
	//af_print(rightSideIndexesZeros);
	array rightSideIndexesN_2 = constant(N / 2, d_NoOfElements, 1, PRECISION_REAL);
	//af_print(rightSideIndexesN_2);

	array indexedElementsEn = join(0, leftSideIndexes, flip(rightSideIndexesOnes, 0));
	array indexedElementsPre = join(0, leftSideIndexes, rightSideIndexesN_2);     /* This is for Keeping pre and post multiplication factor upper half only*/
	array indexedElementsPost = join(0, leftSideIndexes, rightSideIndexesZeros);
	//af_print(indexedElementsEn);
	//af_print(indexedElementsPre_Post);

	 indexedElements_Tiled_En = tile(pow(indexedElementsEn, 2), 1, d_NoOfElements, d_NoOfElements);
	//af_print(indexedElements_Tiled_En.slice(0));
	 indexedElements_Tiled_PreMulti = tile(indexedElementsPre - N / 2, 1, d_NoOfElements, d_NoOfElements);
	//af_print(indexedElements_Tiled_PreMulti.slice(0));
	 indexedElements_Tiled_PostMulti = tile(indexedElementsPost, 1, d_NoOfElements, d_NoOfElements);
	//af_print(indexedElements_Tiled_PostMulti.slice(0));
}


array ComputeNonVectorColumnwise_FrFTUniform(array& Image_3D, double ColumnScale, int d_NoOfElements)
{
	// Image_3D is unpadded just a 3D volume (d_NoOfElements)^3 whose column wise FrFT scaling needs to be done
	// ColumnScale is the given scale that needs to be operated on the image uniformly for entire volume (3D) 
	// d_NoOfElements is the number of elements in the volume Image_3D which is a perfect cube, 


	/**************************************************Preparing the 3D Volume Stack***************************************************/
	array zeros = constant(0, d_NoOfElements, d_NoOfElements, d_NoOfElements, PRECISION_REAL);
	array paddedImage_3D = join(0, Image_3D, zeros);									  // Column Wise padding
	//af_print(paddedImage_3D.slice(0));
	int N = d_NoOfElements - 1;

	/*-------------------------------------------Creating Index Cubes and Sequences----------------------------------------------------*/
	
	//array leftSideIndexes = array(seq(0, N, 1)).as(PRECISION_REAL);
	//// af_print(leftSideIndexes);
	//array rightSideIndexesOnes = -1 * array(seq(1, d_NoOfElements, 1)).as(PRECISION_REAL);
	////af_print(flip( rightSideIndexesOnes,0));
	//array rightSideIndexesZeros = constant(0, d_NoOfElements, 1, PRECISION_REAL);
	////af_print(rightSideIndexesZeros);
	//array rightSideIndexesN_2 = constant(N / 2, d_NoOfElements, 1, PRECISION_REAL);
	////af_print(rightSideIndexesN_2);

	//array indexedElementsEn = join(0, leftSideIndexes, flip(rightSideIndexesOnes, 0));
	//array indexedElementsPre = join(0, leftSideIndexes, rightSideIndexesN_2);     /* This is for Keeping pre and post multiplication factor upper half only*/
	//array indexedElementsPost = join(0, leftSideIndexes, rightSideIndexesZeros);
	////af_print(indexedElementsEn);
	////af_print(indexedElementsPre_Post);

	//array indexedElements_Tiled_En = tile(pow(indexedElementsEn, 2), 1, d_NoOfElements, d_NoOfElements);
	////af_print(indexedElements_Tiled_En.slice(0));
	//array indexedElements_Tiled_PreMulti = tile(indexedElementsPre - N / 2, 1, d_NoOfElements, d_NoOfElements);
	////af_print(indexedElements_Tiled_PreMulti.slice(0));
	//array indexedElements_Tiled_PostMulti = tile(indexedElementsPost, 1, d_NoOfElements, d_NoOfElements);
	////af_print(indexedElements_Tiled_PostMulti.slice(0));


	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	//array ImaginaryUnitTiled = tile(i, 2 * d_NoOfElements, d_NoOfElements, d_NoOfElements);
	//af_print(ImaginaryUnitTiled.slice(0));
	array Seq_En = cexp(complex(0, -Pi * indexedElements_Tiled_En * ColumnScale / d_NoOfElements));   /* E(n) as defined in the paper*/
	//af_print((Pi * indexedElements_Tiled_En * ColumnScale / d_NoOfElements).slice(0));
	//af_print(Seq_En.slice(0));
	//array ones = constant(1, d_NoOfElements, 1, PRECISION_REAL);
	//array subtractValues = tile(join(0, rightSideIndexesZeros, ones), 1, d_NoOfElements, d_NoOfElements);			/* This is for  Keeping pre and post multiplication factor upper half only*/
	array PreMultiplicationFactor = cexp(complex(0, af::Pi * indexedElements_Tiled_PreMulti * ColumnScale * N / d_NoOfElements));// -subtractValues;
	//af_print(PreMultiplicationFactor.slice(0));
	array PostMultiplicationFactor = cexp(complex(0, af::Pi *  indexedElements_Tiled_PostMulti * ColumnScale * N / d_NoOfElements));// - subtractValues;
	//af_print(PostMultiplicationFactor.slice(0));

	/*--------------------Preprocessing Cubes-----------------------*/
	array Image_PreMulti = paddedImage_3D * PreMultiplicationFactor;
	//af_print(Image_PreMulti.slice(0));
	array Image_PreMulti_SeqEn = Image_PreMulti * Seq_En;
	//af_print(Image_PreMulti_SeqEn.slice(0));

	/*-------------------Computing Convolution--------------------*/
	array firstFFT_Image = fft(Image_PreMulti_SeqEn);
	//af_print(firstFFT_Image.slice(0));
	array secondFFT_Seq_En = fft(conjg(Seq_En));
	//af_print(secondFFT_Seq_En.slice(0));
	array interim_FrFT_Image = ifft(firstFFT_Image * secondFFT_Seq_En);
	//af_print(interim_FrFT_Image.slice(0));

	/*-------------------Postprocessing-----------------------------*/
	array  Final_FrFT_Image = interim_FrFT_Image * Seq_En * PostMultiplicationFactor;
	//af_print(Final_FrFT_Image.slice(0));

	/*--------------------Grab only the top half elements drop overlapping------------------*/
	Final_FrFT_Image = Final_FrFT_Image.rows(0, N);
	//af_print(Final_FrFT_Image.slice(0));

	return Final_FrFT_Image;
}

void ComputeNonVector_NonUniformFrFT_SingleColumn(array&FrFT1D_Image_3D, double ColumnScale, int d_NoOfElements, array& Slice2D, array& Slice2D_Conj)
{
	int N = d_NoOfElements - 1;
	array lineSpacing = array(seq(-N / 2, N / 2)).as(PRECISION_REAL);
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	//af_print(lineSpacing_tiled_Y);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
	//af_print(lineSpacing_Square);
	array lineSpacing_Square_TiledLevel = tile(lineSpacing_Square, 1, 1, d_NoOfElements);
	//af_print(lineSpacing_Square_TiledLevel);
	array Column_ScaleFactor = cexp(complex(0, -2 * af::Pi * lineSpacing_Square_TiledLevel * ColumnScale / d_NoOfElements));
	//af_print(Column_ScaleFactor.slice(0))

	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	array realRealPart;             // ac
	array realImagPart;				// ad 
	array imagRealPart;				// bc
	array imagImagPart;				// bd

	SplitMultiplyComplex(FrFT1D_Image_3D, Column_ScaleFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);
	Slice2D = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); // sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor);
	Slice2D_Conj = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart));//  sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor));

	/*af_print(Slice2D);
	af_print(Slice2D_Conj);*/

	Slice2D = moddims(Slice2D, d_NoOfElements, d_NoOfElements);                // Making it 2D without changing the order of data
	Slice2D_Conj = moddims(Slice2D_Conj, d_NoOfElements, d_NoOfElements);

	/*af_print(Slice2D);
	af_print(Slice2D_Conj);*/
}

void ComputeFinal_2ComplementaryLines(array& FrFTFrFT_Image2D, double ColumnScale, int d_NoOfElements, array& Line1D, array& Line1D_Conj)
{
	int N = d_NoOfElements - 1;
	array lineSpacing = array(seq(-N / 2, N / 2)).as(PRECISION_REAL);
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	//af_print(lineSpacing_tiled_Y);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
	array Column_ScaleFactor = cexp(complex(0, -2 * af::Pi * lineSpacing_Square * ColumnScale / d_NoOfElements));
	//af_print(Column_ScaleFactor)

	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	array realRealPart;             // ac
	array realImagPart;				// ad 
	array imagRealPart;				// bc
	array imagImagPart;				// bd

	SplitMultiplyComplex(FrFTFrFT_Image2D, Column_ScaleFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);
	Line1D = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); // sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor);
	Line1D_Conj = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart));//  sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor));
	//af_print(Line1D);
	//af_print(Line1D_Conj);
}

array ComputeForSinglePoint3D(array& Image_3D, array& gridSpacing, int d_NoOfElements, double* desiredPoint)
{
	int N = d_NoOfElements - 1;

	// First since Columns are first dimension in ArrayFire , Y-axis scaling first
	array y_Multiplicationfactor = cexp(complex(0, -2 * Pi*desiredPoint[1] * gridSpacing / d_NoOfElements));

	array Map_y = tile(y_Multiplicationfactor, 1, d_NoOfElements, d_NoOfElements);
	//af_print(Image_3D);
	//af_print(Map_y);
	array Image_2D = moddims(sum(Image_3D*Map_y), d_NoOfElements, d_NoOfElements);
	//af_print(Image_2D);

	// X-axis 

	array x_Multiplicationfactor = cexp(-complex(0, 2 * Pi*desiredPoint[0] * gridSpacing / d_NoOfElements));

	array Map_x = tile(x_Multiplicationfactor, 1, d_NoOfElements);
	//af_print(Map_x);
	array Image_1D = sum(Image_2D*Map_x);
	//af_print(Image_1D);

	// Finally Z-axis 

	array z_Multiplicationfactor = cexp(-complex(0, 2 * Pi*desiredPoint[2] * gridSpacing / d_NoOfElements));
	Image_1D = moddims(Image_1D, d_NoOfElements, 1);
	//af_print(z_Multiplicationfactor);
	//af_print(Image_1D);
	array SinglePoint = sum(Image_1D* z_Multiplicationfactor);
	return SinglePoint;

}

void ComputeDirectBruteForce3D(array& Image_3D, int d_NoOfElements, int d_NoOfAnglesTheta, int d_NoOfAnglesPhi, array& FinalSphericalGrid)
{
	int N = d_NoOfElements - 1;
	array gridSpacing = array(seq(-N / 2, N / 2)).as(PRECISION_REAL);
	array element;
	
	double desiredPoint[3];
	

	for (int p = 0; p < d_NoOfAnglesTheta; ++p)                          // Gives the choice of Polar Slices  in the XY plane
	{
		double angleTheta = p * Pi / d_NoOfAnglesTheta;
		for (int q = 0; q < d_NoOfAnglesPhi; ++q)                       // Gives the choice of Anglular lines measured from the XY plane
		{
			double anglePhi = q * Pi / d_NoOfAnglesPhi;

			//timer::start();
			for (int n = 0; n < d_NoOfElements; ++n)                    // Gives choices of points on the angular lines
			{
				element = gridSpacing.row(n);						   // Radial distance
				double rho = element.scalar<double>();

				desiredPoint[0] = rho * cos(angleTheta)*cos(anglePhi);    // x coordinate 
				desiredPoint[1] = rho * cos(angleTheta)*sin(anglePhi);   // y coordinate 
				desiredPoint[2] = rho * sin(angleTheta);					 // z coordinate 

				//desiredPoint[0] = rho * cos(anglePhi)*cos(angleTheta);   Previous // x coordinate 
				//desiredPoint[1] = rho * cos(anglePhi)*sin(angleTheta);   // y coordinate 
				//desiredPoint[2] = rho * sin(anglePhi);					 // z coordinate 

				FinalSphericalGrid(p, q, n) = ComputeForSinglePoint3D(Image_3D, gridSpacing, d_NoOfElements, desiredPoint);
			}
			//printf("elapsed seconds: %g at iteration %d\n", timer::stop(),q);
		}
		
	}
}

array Get2DFullPolarDFT(array& Image, int d_NoOfElements, int d_NoOfAngles, int d_NoOfLevels)    // This will require the same processing as defined for 2D see the solution in 2D
{
	
	int N = d_NoOfElements - 1;
	cdouble i_cdouble = { 0, 1 };
	array i = constant(i_cdouble, 1, 1, PRECISION_COMPLEX);/* imaginary unit */

	/*-----------------------------------Preparing Padded & Tiled Images in X and Y--------------------------------------------*/
	//array Zeros = constant(0, d_NoOfElements, d_NoOfElements, PRECISION_REAL);
	array Zeros = constant(0, d_NoOfElements, d_NoOfElements, PRECISION_COMPLEX);
	array Ones = constant(1, d_NoOfElements, d_NoOfElements, PRECISION_COMPLEX);
	//af_print(Image);
	//af_print(Zeros);
	array Zero_Padded_X_Image_Transposed = join(1, Image, Zeros).T();        /* Transposing so that the rows acts as columns to be operated upon for FFTs*/
	array Zero_Padded_Y_Image = join(0, flip(Image, 0), Zeros);              /* Needed since image gathered gives +ve to -ve indexes and we require opposite for computations*/
	array Image_Tiled_X = tile(Zero_Padded_X_Image_Transposed, 1, 1, d_NoOfLevels);
	array Image_Tiled_Y = tile(Zero_Padded_Y_Image, 1, 1, d_NoOfLevels);

	/*-------------------------------------------Creating Index Cubes and Sequences----------------------------------------------------*/
	array leftSideIndexes = array(seq(0, N, 1)).as(PRECISION_REAL);
	array rightSideIndexesOnes = -1 * array(seq(1, d_NoOfElements, 1)).as(PRECISION_REAL);
	array rightSideIndexesZeros = constant(0, d_NoOfElements, 1, PRECISION_REAL);
	array rightSideIndexesN_2 = constant(N / 2, d_NoOfElements, 1, PRECISION_REAL);

	array indexedElementsEn = join(0, leftSideIndexes, flip(rightSideIndexesOnes, 0));
	array indexedElementsPre = join(0, leftSideIndexes, rightSideIndexesN_2);     /* This is for Keeping pre and post multiplication factor upper half only*/
	array indexedElementsPost = join(0, leftSideIndexes, rightSideIndexesZeros);

	array indexedElements_Tiled_En = tile(pow(indexedElementsEn, 2), 1, d_NoOfElements, d_NoOfLevels);
	array indexedElements_Tiled_PreMulti = tile(indexedElementsPre - N / 2, 1, d_NoOfElements, d_NoOfLevels);
	array indexedElements_Tiled_PostMulti = tile(indexedElementsPost, 1, d_NoOfElements, d_NoOfLevels);

	array lineSpacing = array(seq(-N / 2, N / 2)).as(PRECISION_REAL);
	array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
	array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
	array lineSpacing_Square_TiledLevel = tile(lineSpacing_Square, 1, 1, d_NoOfLevels);
	
	/*--------------------------Creating Alpha Cubes------------------------------------*/
	array alpha_Levels = cos(af::Pi / d_NoOfAngles * array(seq(1, d_NoOfLevels)).as(PRECISION_REAL));
	array alpha_Mods = moddims(alpha_Levels, 1, 1, d_NoOfLevels);
	array alpha_Tiled_depth = tile(alpha_Mods, 2 * d_NoOfElements, d_NoOfElements, 1);
	array alpha_Tiled = moddims(alpha_Tiled_depth, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels);


	/*--------------------------Creating Beta Cubes---------------------------------------*/
	array beta_Levels = sin(Pi / d_NoOfAngles * array(seq(1, d_NoOfLevels)).as(PRECISION_REAL));
	array beta_Mods = moddims(beta_Levels, 1, 1, d_NoOfLevels);
	array beta_Tiled_depth = tile(beta_Mods, d_NoOfElements, d_NoOfElements, 1);
	array beta_Tiled = moddims(beta_Tiled_depth, d_NoOfElements, d_NoOfElements, d_NoOfLevels);
	

	/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
	array Seq_En = cexp(-tile(i, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels) * af::Pi * indexedElements_Tiled_En * alpha_Tiled / d_NoOfElements);   /* E(n) as defined in the paper*/
	
	array subtractValues = tile(join(0, Zeros, Ones), 1, 1, d_NoOfLevels);			/* This is for  Keeping pre and post multiplication factor upper half only*/
	array PreMultiplicationFactor = cexp(tile(i, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels) * af::Pi * indexedElements_Tiled_PreMulti * alpha_Tiled * N / d_NoOfElements) - subtractValues;
	array PostMultiplicationFactor = cexp(tile(i, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels)* af::Pi *  indexedElements_Tiled_PostMulti * alpha_Tiled * N / d_NoOfElements) - subtractValues;
	array BetaFactor = cexp(-2 * tile(i, d_NoOfElements, d_NoOfElements, d_NoOfLevels) * af::Pi * lineSpacing_Square_TiledLevel * beta_Tiled / d_NoOfElements);

	array ZeroNinty_Factor = cexp(-2 * tile(i, d_NoOfElements, d_NoOfElements) * af::Pi * lineSpacing_Square * 1 / d_NoOfElements);

	/*--------------------Preprocessing Cubes-----------------------*/
	array Image_Tiled_X_PreMulti = Image_Tiled_X * PreMultiplicationFactor;
	array Image_Tiled_X_PreMulti_SeqEn = Image_Tiled_X_PreMulti * Seq_En;
	array Image_Tiled_Y_PreMulti = Image_Tiled_Y * PreMultiplicationFactor;
	array Image_Tiled_Y_PreMulti_SeqEn = Image_Tiled_Y_PreMulti * Seq_En;


	/*-------------------Computing Convolution--------------------*/
	int normalizationFactor = pow(2 * d_NoOfElements, 0);			// This is a scalar single value especially needed only for ArrayFire ifft
	array firstFFT_X = fft(moddims(Image_Tiled_X_PreMulti_SeqEn, 2 * d_NoOfElements, d_NoOfLevels*d_NoOfElements));
	array secondFFT_X = fft(moddims(conjg(Seq_En), 2 * d_NoOfElements, d_NoOfLevels*d_NoOfElements));
	array interim_FrFT_X = ifft(firstFFT_X * secondFFT_X) / normalizationFactor;
	array firstFFT_Y = fft(moddims(Image_Tiled_Y_PreMulti_SeqEn, 2 * d_NoOfElements, d_NoOfLevels*d_NoOfElements));
	array secondFFT_Y = secondFFT_X;
	array interim_FrFT_Y = ifft(firstFFT_Y * secondFFT_Y) / normalizationFactor;


	interim_FrFT_X = moddims(interim_FrFT_X, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels);    /* Rearranging matrix to level based volume*/
	interim_FrFT_Y = moddims(interim_FrFT_Y, 2 * d_NoOfElements, d_NoOfElements, d_NoOfLevels);

	/*-------------------Postprocessing-----------------------------*/
	array  FrFT_Image_X = interim_FrFT_X * Seq_En * PostMultiplicationFactor;
	array  FrFT_Image_Y = interim_FrFT_Y * Seq_En * PostMultiplicationFactor;


	/*--------------------Grab only the top half elements drop overlapping------------------*/
	array FrFT_Image_X_Cube = FrFT_Image_X.rows(0, N);
	array FrFT_Image_Y_Cube = FrFT_Image_Y.rows(0, N);

	/*--------------------Finally all computations for  the  Polar Grid-----------*/
	//   Computing for all the grid expect two special indexes
	array levelSeq = array(seq(0, d_NoOfLevels - 1)).as(PRECISION_REAL);
	array finalIndexSeq1_X = 1 + levelSeq;
	array finalIndexSeq2_X = d_NoOfAngles - finalIndexSeq1_X;
	array finalIndexSeq3_Y = d_NoOfAngles / 2 - finalIndexSeq1_X;
	array finalIndexSeq4_Y = d_NoOfAngles / 2 + finalIndexSeq1_X;

	/* This transpose is Required since we will no operate in the column wise transposed axis */
	FrFT_Image_X_Cube = FrFT_Image_X_Cube.T();
	FrFT_Image_Y_Cube = FrFT_Image_Y_Cube.T();

	// Consider multiplication of complex numbers A = (a+ib); B = (c+id), This is used to reduce the computations by half 	
	array realRealPart;             // ac
	array realImagPart;				// ad
	array imagRealPart;				// bc
	array imagImagPart;				// bd

	SplitMultiplyComplex(flip(FrFT_Image_X_Cube, 0), BetaFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);
	array tempSeq_X = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); // sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor);
	array tempSeqConj_X = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart));//  sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor));
	SplitMultiplyComplex(FrFT_Image_Y_Cube, BetaFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);
	array tempSeq_Y = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); //sum(FrFT_Image_Y_Cube * BetaFactor);
	array tempSeqConj_Y = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart)); //sum(FrFT_Image_Y_Cube *conjg(BetaFactor));

	array finalSeq_X = moddims(tempSeq_X, d_NoOfElements, d_NoOfLevels).T();
	array finalSeqConj_X = moddims(tempSeqConj_X, d_NoOfElements, d_NoOfLevels).T();
	finalSeqConj_X = flip(finalSeqConj_X, 1);
	array finalSeq_Y = moddims(tempSeq_Y, d_NoOfElements, d_NoOfLevels).T();
	array finalSeqConj_Y = moddims(tempSeqConj_Y, d_NoOfElements, d_NoOfLevels).T();

	// Removing just 2 redundant computations for 45 degree case
	if (0 == remainder(d_NoOfAngles, 4))
	{
		finalIndexSeq3_Y = finalIndexSeq3_Y.rows(0, d_NoOfLevels - 2);          // Removing just the last rows from 4 structures
		finalSeq_Y = finalSeq_Y.rows(0, d_NoOfLevels - 2);
		finalIndexSeq4_Y = finalIndexSeq4_Y.rows(0, d_NoOfLevels - 2);
		finalSeqConj_Y = finalSeqConj_Y.rows(0, d_NoOfLevels - 2);
	}

	//   Computing seperately for two special indexes
	array SpecialTwoIndexes;
	array ZeroLineFrFT_Image_X_Cube;
	array NintyLineFrFT_Image_Y_Cube;
	if (PRECISION_REAL == f32)
	{
	   float zeroIndex = 0;
	   float nintyIndex = d_NoOfAngles / 2;
	    float values[] = { zeroIndex, nintyIndex };
	   SpecialTwoIndexes= array(2, 1, values);
	   ZeroLineFrFT_Image_X_Cube = FrFT_Image_Y_Cube.slice(zeroIndex).col(N / 2);
	   NintyLineFrFT_Image_Y_Cube = FrFT_Image_X_Cube.slice(zeroIndex).col(N / 2);
	}
	else
	{
		double zeroIndex = 0;
		double nintyIndex = d_NoOfAngles / 2;
		double values[] = { zeroIndex, nintyIndex };
		SpecialTwoIndexes = array(2, 1, values);
		ZeroLineFrFT_Image_X_Cube = FrFT_Image_Y_Cube.slice(zeroIndex).col(N / 2);
		NintyLineFrFT_Image_Y_Cube = FrFT_Image_X_Cube.slice(zeroIndex).col(N / 2);
	}

	

	array DFTZeroLine = sum(tile(ZeroLineFrFT_Image_X_Cube, 1, d_NoOfElements) *ZeroNinty_Factor);
	array DFTNinetyLine = sum(tile(flip(NintyLineFrFT_Image_Y_Cube, 0), 1, d_NoOfElements) *ZeroNinty_Factor);
	array SpecialTwoLines = join(0, DFTZeroLine, DFTNinetyLine);


	array UnsortedIndexes = join(0, join(0, join(0, join(0, finalIndexSeq1_X, finalIndexSeq2_X), finalIndexSeq3_Y), finalIndexSeq4_Y), SpecialTwoIndexes);
	array tiledUnsortedIndexes = tile(UnsortedIndexes, 1, d_NoOfElements);
	array UnsortedPolarGrid = join(0, join(0, join(0, join(0, finalSeq_X, finalSeqConj_X), finalSeq_Y), finalSeqConj_Y), SpecialTwoLines);


	array FinalPolarGridReal;// = constant(0, d_NoOfElements, d_NoOfAngles, PRECISION_COMPLEX);
	array Output_Keys_Sorted;
	sort(Output_Keys_Sorted, FinalPolarGridReal, tiledUnsortedIndexes, real(UnsortedPolarGrid));

	array FinalPolarGridImag;// = constant(0, d_NoOfElements, d_NoOfAngles, PRECISION_COMPLEX);
	array Output_Keys_Sorted2;
	sort(Output_Keys_Sorted2, FinalPolarGridImag, tiledUnsortedIndexes, imag(UnsortedPolarGrid));


	return complex(FinalPolarGridReal, FinalPolarGridImag);
}

static void ComputeFastSphericalPolarFourierTransform(array& Image3D, array& FinalSphericalGrid, int d_NoOfElements, int d_NoOfAnglesTheta, int d_NoOfLevelsTheta, int d_NoOfAnglesPhi, int d_NoOfLevelsPhi)
{
	array ReorderedImage_OperateColumns;                 // Reordered 3D Image at the start of the FrFT operation  
	array FrFT1D_Uniform_Image3D;                        // 3D Image obtained after passing 1st stage of FrFT, all columns have now been operated on
	array Silce2D, Slice2D_Conj;                         // 2D Slices obtained at the 2nd stage of FrFT
	array Line1, Line1_Conj, Line2, Line2_Conj;			 // 1D Lines at the end of the 3rd stage of FrFT, which are final	
	double alpha_factor, beta_factor, gamma_factor;      // Scaling factors in X-axis , Y-axis and Z-axis respectively, which change depending on the computation of the block 
	
	int N = d_NoOfElements - 1;
	GlobalArraysComputeInitialize(d_NoOfElements);
	for (int q = 1; q <= d_NoOfLevelsPhi; ++q)                // Splitting as defined in the paper (See the schematic of spherical divisions)
	{
		double anglePhi = q * Pi / d_NoOfAnglesPhi;

		//timer::start();
		for (int p = 1; p <= d_NoOfLevelsTheta; ++p)       // X oriented pair of Polar slices, ( q, d_NoOfAnglesPhi - q)
		{
			double angleTheta = p  * Pi / d_NoOfAnglesTheta;

			/**************************************************************************************************************************************************************************/
			// XX block  -- Concentric rectangles in YZ tiled along X -axis
			alpha_factor = cos(angleTheta) * cos(anglePhi);											         // Scaling needed as defined in the paper
			beta_factor = cos(angleTheta) * sin(anglePhi);
			gamma_factor = sin(angleTheta);

			ReorderedImage_OperateColumns = reorder(Image3D, X_AXIS, Y_AXIS, Z_AXIS);                                //  1st Swap, X-axis as columns now
			//af_print(Image3D.T().slices(0, 2));
			//af_print(ReorderedImage_OperateColumns.slices(0,2));
			//ReorderedImage_OperateColumns = reorder(ReorderedImage_OperateColumns, X_AXIS, Y_AXIS, Z_AXIS );            // Restoring the image as it was
			//af_print(ReorderedImage_OperateColumns.slices(0,2));

			FrFT1D_Uniform_Image3D = ComputeNonVectorColumnwise_FrFTUniform(ReorderedImage_OperateColumns, alpha_factor, d_NoOfElements); // This is the most expensive operation of order (N+1)^3 log(N+1)
			//af_print(FrFT1D_Uniform_Image3D)
			if (q == 1 && p == 1)         // Computing Polar Slice at anglephi = 90 , only need to be computed once
			{
				array Central2D_YZSlice2D = FrFT1D_Uniform_Image3D(N / 2, span, span);
				Central2D_YZSlice2D = moddims(Central2D_YZSlice2D, d_NoOfElements, d_NoOfElements);
				//af_print(Central2D_YZSlice2D);

				array properOrientedSlice = flip(Central2D_YZSlice2D.T(), 0);          // Verified match 
				//af_print(properOrientedSlice);
				array Polar2D = Get2DFullPolarDFT(properOrientedSlice, d_NoOfElements, d_NoOfAnglesTheta, d_NoOfLevelsTheta);
				//af_print(FinalSphericalGrid(span, d_NoOfAnglesPhi / 2, span));
				Polar2D = moddims(Polar2D, d_NoOfAnglesTheta, 1, d_NoOfElements);
				//af_print(Polar2D);
				FinalSphericalGrid(span, d_NoOfAnglesPhi / 2, span) = Polar2D;
				//af_print(BruteForceSphericalGrid(span, d_NoOfAnglesPhi / 2, span) - FinalSphericalGrid(span, d_NoOfAnglesPhi / 2, span));
			}

			//af_print(FrFT1D_Uniform_Image3D.slice(0))
			//af_print((reorder(FrFT1D_Uniform_Image3D, Z_AXIS, Y_AXIS, X_AXIS)).slice(0))
			//af_print(FrFT1D_Uniform_Image3D);
			//af_print(reorder(FrFT1D_Uniform_Image3D, Y_AXIS, X_AXIS, Z_AXIS));
			ComputeNonVector_NonUniformFrFT_SingleColumn(reorder(FrFT1D_Uniform_Image3D, X_AXIS, Y_AXIS, Z_AXIS), beta_factor, d_NoOfElements, Silce2D, Slice2D_Conj); //  2nd Swap, Y-axis as columns
			//af_print(Silce2D);
			//af_print(Slice2D_Conj);

			// Gather 4 lines
			ComputeFinal_2ComplementaryLines(Silce2D.T(), gamma_factor, d_NoOfElements, Line1, Line1_Conj);          // 3rd Swap , Z axis as columns
			ComputeFinal_2ComplementaryLines(Slice2D_Conj.T(), gamma_factor, d_NoOfElements, Line2, Line2_Conj);      // 3rd Swap , Z axis as columns

			//af_print(Line1);
			//af_print(Line1_Conj); 
			//af_print(Line2);
			//af_print(Line2_Conj);

			FinalSphericalGrid(p, q, span) = Line1;
			FinalSphericalGrid(p, d_NoOfAnglesPhi - q, span) = conjg(Line2_Conj);
			FinalSphericalGrid(d_NoOfAnglesTheta - p, q, span) = conjg(Line1_Conj);											// 1 More Swap to match !
			FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi - q, span) = Line2;

			/*printf("\n My Special Solution 1st XX block\n");
			af_print(moddims(FinalSphericalGrid(p, q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(p, d_NoOfAnglesPhi - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta - p, q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi - q, span), 1, d_NoOfElements));
			printf("\n");*/
			/**************************************************************************************************************************************************************************/

			if (p * 180 / d_NoOfAnglesTheta == 45)       // At 45 degrees its redundant to compute in the next block so skip it
				continue;


			/**************************************************************************************************************************************************************************/
			// XZ block  -- Concentric rectangles in XY tiled along Z -axis
			alpha_factor = sin(angleTheta) * cos(anglePhi);											         // Scaling needed as defined in the paper
			beta_factor = sin(angleTheta) * sin(anglePhi);
			gamma_factor = cos(angleTheta);

			ReorderedImage_OperateColumns = reorder(Image3D, Z_AXIS, Y_AXIS, X_AXIS);                                   //  1st Swap, Z-axis elements in columns now
			//af_print(ReorderedImage_OperateColumns.slices(0,2));
			//ReorderedImage_OperateColumns = reorder(ReorderedImage_OperateColumns, X_AXIS, Z_AXIS, Y_AXIS );           // Restoring the image as it was
			//af_print(ReorderedImage_OperateColumns.slices(0, 2));
			FrFT1D_Uniform_Image3D = ComputeNonVectorColumnwise_FrFTUniform(ReorderedImage_OperateColumns, gamma_factor, d_NoOfElements); // This is the most expensive operation of order (N+1)^3 log(N+1)
			//af_print(FrFT1D_Uniform_Image3D);

			if (q == 1 && p == 1)         // Computing One special Polar Slice at angletheta = 0 , only need to be computed once
			{
				array Central2D_XYSlice2D = FrFT1D_Uniform_Image3D(N / 2, span, span);
				Central2D_XYSlice2D = moddims(Central2D_XYSlice2D, d_NoOfElements, d_NoOfElements);
				//af_print(Central2D_XYSlice2D);

				array properOrientedSlice = flip(Central2D_XYSlice2D, 0);          // Verified match 
				array Polar2D = Get2DFullPolarDFT(properOrientedSlice, d_NoOfElements, d_NoOfAnglesPhi, d_NoOfLevelsPhi);
				Polar2D = moddims(Polar2D, 1, d_NoOfAnglesPhi, d_NoOfElements);
				//af_print(Polar2D);
				FinalSphericalGrid(0, span, span) = Polar2D;
				//af_print(BruteForceSphericalGrid(0, span, span) - FinalSphericalGrid(0, span, span));
			}


			ComputeNonVector_NonUniformFrFT_SingleColumn(FrFT1D_Uniform_Image3D.T(), beta_factor, d_NoOfElements, Silce2D, Slice2D_Conj); //  2nd Swap, Y-axis as columns

			// Gather 4 lines
			ComputeFinal_2ComplementaryLines(Silce2D.T(), alpha_factor, d_NoOfElements, Line1, Line1_Conj);          // 3rd Swap , X axis as columns
			ComputeFinal_2ComplementaryLines(Slice2D_Conj.T(), alpha_factor, d_NoOfElements, Line2, Line2_Conj);     // Swap Lines ! // 3rd Swap , Z axis as columns

			FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, q, span) = Line1;
			FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi - q, span) = (Line1_Conj);

			FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, q, span) = (Line2_Conj);
			FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi - q, span) = (Line2);

			/*printf("\n My Special Solution 1st XZ block\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi - q, span), 1, d_NoOfElements));
			printf("\n");*/

			/**************************************************************************************************************************************************************************/

			if (q * 180 / d_NoOfAnglesPhi == 45)       // At 45 degrees its redundant to compute same in the next block  so skip it
				continue;

			/**************************************************************************************************************************************************************************/
			// YZ block                                                   // Y oriented pair of Polar slices, (d_NoOfAnglesPhi / 2 - q, d_NoOfAnglesPhi / 2 + q)
			alpha_factor = sin(angleTheta) * sin(anglePhi);				 // Scaling needed as defined in the paper
			beta_factor = sin(angleTheta) * cos(anglePhi);
			gamma_factor = cos(angleTheta);

			// Reusing the previously computed FrFT 1D Uniform scaling since the gamma_factor for both XZ and YZ block is the same

			//ReorderedImage_OperateColumns = reorder(Image3D, Z_AXIS, Y_AXIS, X_AXIS);                                   //  1st Swap, Z-axis elements in columns now
			//FrFT1D_Uniform_Image3D = ComputeNonVectorColumnwise_FrFTUniform(ReorderedImage_OperateColumns, gamma_factor, d_NoOfElements); // This is the most expensive operation of order (N+1)/4*(N+1)^3 log(N+1)
			ComputeNonVector_NonUniformFrFT_SingleColumn(FrFT1D_Uniform_Image3D.T(), beta_factor, d_NoOfElements, Silce2D, Slice2D_Conj); //  2nd Swap, Y-axis as columns

			// Gather 4 lines
			ComputeFinal_2ComplementaryLines(Silce2D.T(), alpha_factor, d_NoOfElements, Line1, Line1_Conj);          // 3rd Swap , X axis as columns
			ComputeFinal_2ComplementaryLines(Slice2D_Conj.T(), alpha_factor, d_NoOfElements, Line2, Line2_Conj);     // Swap Lines ! // 3rd Swap , Z axis as columns

			FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi / 2 - q, span) = Line1;
			FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi / 2 + q, span) = Line1_Conj;

			FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi / 2 - q, span) = Line2_Conj;
			FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi / 2 + q, span) = Line2;

			/*printf("\n My Special Solution 1st YZ block\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi / 2 - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 - p, d_NoOfAnglesPhi / 2 + q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi / 2 - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta / 2 + p, d_NoOfAnglesPhi / 2 + q, span), 1, d_NoOfElements));
			printf("\n");*/
			/**************************************************************************************************************************************************************************/

		}
		//printf("elapsed seconds: %g at iteration %d X-oriented Polar Slices\n", timer::stop(),q);

		if (q * 180 / d_NoOfAnglesPhi == 45)       // At 45 degrees its redundant to compute same in the next block  so skip it
			continue;
		//timer::start();

		for (int p = 1; p <= d_NoOfLevelsTheta; ++p)        // Y oriented pair of Polar slices, (d_NoOfAnglesPhi / 2 - q, d_NoOfAnglesPhi / 2 + q)
		{
			double angleTheta = p  * Pi / d_NoOfAnglesTheta;

			/**************************************************************************************************************************************************************************/
			// YY block
			alpha_factor = cos(angleTheta) * sin(anglePhi);				 // Scaling needed as defined in the paper
			beta_factor = cos(angleTheta) * cos(anglePhi);
			gamma_factor = sin(angleTheta);

			ReorderedImage_OperateColumns = Image3D;                                         // No swap directly we can operate on columns
			FrFT1D_Uniform_Image3D = ComputeNonVectorColumnwise_FrFTUniform(ReorderedImage_OperateColumns, beta_factor, d_NoOfElements); // This is the most expensive operation of order (N+1)^3 log(N+1)

			if (q == 1 && p == 1)         // Computing Polar Slice at anglephi = 0, only need to be computed once
			{ 
				array Central2D_YZSlice2D = FrFT1D_Uniform_Image3D(N / 2, span, span);
				Central2D_YZSlice2D = moddims(Central2D_YZSlice2D, d_NoOfElements, d_NoOfElements);
				//af_print(Central2D_YZSlice2D);

				array properOrientedSlice = flip(Central2D_YZSlice2D.T(), 0);          // Verified match  
				array Polar2D = Get2DFullPolarDFT(properOrientedSlice, d_NoOfElements, d_NoOfAnglesTheta, d_NoOfLevelsTheta);
				Polar2D = moddims(Polar2D, d_NoOfAnglesTheta, 1, d_NoOfElements);
				//af_print(Polar2D);
				FinalSphericalGrid(span, 0, span) = Polar2D;
				//af_print(BruteForceSphericalGrid(span, 0, span) - FinalSphericalGrid(span, 0, span));


				// Special Operation
				//af_print(Polar2D(d_NoOfAnglesTheta / 2, span));
				array SpecialLineZ = Polar2D(d_NoOfAnglesTheta / 2, span);
				//af_print(SpecialLineZ);
				array TiledZ = tile(SpecialLineZ, 1, d_NoOfAnglesPhi, 1);
				//af_print(TiledZ);
				//af_print(FinalSphericalGrid(d_NoOfAnglesTheta / 2, span, span));
				FinalSphericalGrid(d_NoOfAnglesTheta / 2, span, span) = TiledZ;        // This line is common to all grids !!! VERY VERY Special
			}

			ComputeNonVector_NonUniformFrFT_SingleColumn(FrFT1D_Uniform_Image3D.T(), alpha_factor, d_NoOfElements, Silce2D, Slice2D_Conj); //  2nd Swap, X-axis as columns now

			// Gather 4 lines
			ComputeFinal_2ComplementaryLines(Silce2D.T(), gamma_factor, d_NoOfElements, Line1, Line1_Conj);          // 3rd Swap , Z axis as columns
			ComputeFinal_2ComplementaryLines(Slice2D_Conj.T(), gamma_factor, d_NoOfElements, Line2, Line2_Conj);     // Swap Lines ! // 3rd Swap , Z axis as columns


			FinalSphericalGrid(p, d_NoOfAnglesPhi / 2 - q, span) = Line1;
			FinalSphericalGrid(p, d_NoOfAnglesPhi / 2 + q, span) = Line2;

			FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi / 2 - q, span) = conjg(Line1_Conj);
			FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi / 2 + q, span) = conjg(Line2_Conj);

			/*printf("\n My Special Solution 1st YY block\n");
			af_print(moddims(FinalSphericalGrid(p, d_NoOfAnglesPhi / 2 - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(p, d_NoOfAnglesPhi / 2 + q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi / 2 - q, span), 1, d_NoOfElements));
			printf("\n");
			af_print(moddims(FinalSphericalGrid(d_NoOfAnglesTheta - p, d_NoOfAnglesPhi / 2 + q, span), 1, d_NoOfElements));
			printf("\n");*/
			/**************************************************************************************************************************************************************************/

			//if (p * 180 / d_NoOfAnglesTheta == 45)       // At 45 degrees its redundant to compute in the next block       so skip it
			//	continue;

			
		}
		//printf("elapsed seconds: %g at iteration %d Y-oriented Polar Slices\n", timer::stop(),q);
	}
	
}

static int d_NoOfElements = 9;		/* IT is always odd */

static void bench()
{
	array Image3D = randu(d_NoOfElements, d_NoOfElements, d_NoOfElements, PRECISION_REAL);
	int d_NoOfAnglesTheta = d_NoOfElements + 13;       /*Always even*/
	int d_NoOfAnglesPhi = d_NoOfElements + 29;        /*Always even*/
	int d_NoOfLevelsTheta = ceil(float(d_NoOfAnglesTheta - 2) / 4);          // This is where partitioning begins
	int d_NoOfLevelsPhi = ceil(float(d_NoOfAnglesPhi - 2) / 4);

	array FinalSphericalGrid = constant(0, d_NoOfAnglesTheta, d_NoOfAnglesPhi, d_NoOfElements, PRECISION_COMPLEX);      // Angle Phi vs. Polar slices : Each Polar slice has No of elevation angle Theta vs. Radial data
	ComputeFastSphericalPolarFourierTransform(Image3D, FinalSphericalGrid, d_NoOfElements, d_NoOfAnglesTheta, d_NoOfLevelsTheta, d_NoOfAnglesPhi, d_NoOfLevelsPhi);

}

static void MakeFileEntriesAsInputs()
{
	std::ofstream  myfile;
	myfile.open("Input.csv");
	//myfile << "InputArguments\n";
	for (int noOfElements = 15; noOfElements < 168; noOfElements = noOfElements + 2)
	{
		myfile << "{Back}{Back}{Back}" << noOfElements <<  "\n";
	}
	myfile.close();
}

static void MakeFileEntriesAsResults(int noOfElements)
{
	std::ofstream  myfile;
	myfile.open("Results.csv", std::ios::app);
	//myfile << "Timing with microsecond accurate clock (GPU) in seconds\n \n \n ";
	//myfile << "(N+1), times(seconds)\n";
	
	
	//for (int noOfElements = 15; noOfElements < 178; noOfElements = noOfElements + 2)
	{
		af::deviceGC();
		d_NoOfElements = noOfElements;
		printf("%dx%dx%d random number generated 3D Image and its Spherical Polar FFT \n", d_NoOfElements, d_NoOfElements, d_NoOfElements);
		double time_s = timeit(bench); // seconds
		printf("Timing with microsecond accurate clock (GPU) %f seconds\n", time_s);
		myfile << d_NoOfElements << "," << time_s << "\n";
	}

	myfile.close();
}



BOOL KillProcessByName(char *szProcessToKill){
	HANDLE hProcessSnap;
	HANDLE hProcess;
	PROCESSENTRY32 pe32;
	DWORD dwPriorityClass;

	hProcessSnap = CreateToolhelp32Snapshot(TH32CS_SNAPPROCESS, 0);  // Takes a snapshot of all the processes

	if (hProcessSnap == INVALID_HANDLE_VALUE){
		return(FALSE);
	}

	pe32.dwSize = sizeof(PROCESSENTRY32);

	if (!Process32First(hProcessSnap, &pe32)){
		CloseHandle(hProcessSnap);
		return(FALSE);
	}

	do{
		if (!strcmp(pe32.szExeFile, szProcessToKill)){    //  checks if process at current position has the name of to be killed app
			hProcess = OpenProcess(PROCESS_TERMINATE, 0, pe32.th32ProcessID);  // gets handle to process
			TerminateProcess(hProcess, 0);   // Terminate process by handle
			CloseHandle(hProcess);  // close the handle
		}
	} while (Process32Next(hProcessSnap, &pe32));  // gets next member of snapshot

	CloseHandle(hProcessSnap);  // closes the snapshot handle
	return(TRUE);
}


 
int main(int argc, char* argv[]) 
{
	try {
		  
		info();
		//int total_device = devicecount();
		//printf("This computer has device count %d \n", total_device);

		//MakeFileEntriesAsInputs();
		//int noOfElements;
		//sscanf(argv[1], "%d", &noOfElements);
		//MakeFileEntriesAsResults(noOfElements);
		
		//KillProcessByName("cmd.exe");
		
		/******************************* Check the result ! *************************************************/
		/***************************************************************************************************/
		/***************************************************************************************************/
		/***************************************************************************************************/
		/* With d_NoOfElements = 129 which seems moderate size of 3D image, this is the result of computations: 
		ArrayFire v3.0.0 (CUDA, 64 - bit Windows, build 86426db17
		Platform: CUDA Toolkit 7, Driver : 0.00
		[0] GeForce GTX 760, 2048 MB, CUDA Compute 3.0
		This computer has device count 1
		129x129x129 random number generated 3D Image and its Spherical Polar FFT --- These timings are only for CPU sides, accurate timing with GPU clock is lower 
		Total time with brute force O((N + 1) ^ 6), elapsed seconds : 14667.2                                     --- This is approx 4 hours !!!
		Total time for my Special solution O(3 x K/4 x M/4 x (N+1)^3/2 x log2(N+1)), elapsed seconds : 410        --- This is approx 6.83 mins !!!!!!!!!!!!

		/*******************************************************************************************************
		with GeForce GTX Titan X we have ...
		Total time for my Special solution O(3 x K/4 x M/4 x (N+1)^3/2 x log2(N+1)), elapsed seconds : 254       --- This is approx 4.233 mins !!!!!!!!!!!!

		For d_NoOfElements = 133, elapsed seconds = 419 seconds
		Error between the direct brute force computation and the fast exact solution
		abs(sum(sum(sum(BruteForceSphericalGrid - FinalSphericalGrid))))[1 1 1 1]
		0.0
		hit[enter]...   */
		/***************************************************************************************************/
		/***************************************************************************************************/
		/***************************************************************************************************/

		 
		/******************************Create DATA********************************************/
		 
		int total_device = devicecount();
		printf("This computer has device count %d \n", total_device);
		printf("%dx%dx%d random number generated 3D Image and its Spherical Polar FFT \n", d_NoOfElements, d_NoOfElements, d_NoOfElements);
		 
		array Image3D(d_NoOfElements, d_NoOfElements, d_NoOfElements, ThreeDImageValues);   // Gather image from the stored values  for testing purposes
//		array Image3D = randu(d_NoOfElements, d_NoOfElements, d_NoOfElements,  PRECISION_REAL);
		Image3D = Image3D.T();
		af_print(Image3D(span,span,0));

		/*****************Learning to display 3 axis data*****************************/
		printf("This is Y -axis !\n");
		af_print(Image3D.slice(0).col(0));			// Y-Axis
		printf("This is X -axis !\n");
		af_print(Image3D.slice(0).row(0));			// X-axis
		printf("This is Z -axis !\n");
		af_print(Image3D(0, 0, span));                // Z-axis
		printf("Successfully completed the computations of 3D DFT on a full Polar Grid !\n");

		/*********************************Processing FFTs along a given axis ************************************************/
		array FFT_Y = fft(Image3D);
		printf("This is Y -axis FFTs !\n");
		af_print(fft(Image3D.slice(0).col(0)));			       // Y-axis
		af_print(FFT_Y.slice(0).col(0))

		array ReorderedImage = reorder(Image3D, X_AXIS, Y_AXIS, Z_AXIS);
		af_print(ReorderedImage.slices(1, 3));
		array TransposedImage3d = Image3D.T();
		af_print(TransposedImage3d.slices(1, 3));
		array FFT_X = fft(TransposedImage3d);
		printf("This is X -axis FFTs !\n");
		af_print(fft(TransposedImage3d.slice(0).col(0)));		// X-axis
		af_print(FFT_X.slice(0).col(0))

		array ZAxis_TransposedImage3D = reorder(Image3D, 2, 0, 1) ;
		array FFT_Z = fft(ZAxis_TransposedImage3D);
		printf("This is Z -axis FFTs !\n");
		af_print(fft(ZAxis_TransposedImage3D.slice(0).col(0)));		// X-axis
		af_print(FFT_Z.slice(0).col(0))
		 
		int N = d_NoOfElements - 1; /* it is always even as described in the paper*/
		int d_NoOfAnglesTheta = d_NoOfElements + 1;       /*Always even*/
		int d_NoOfAnglesPhi = d_NoOfElements + 5;        /*Always even*/

		
		/***************************          Brute Force solution  order   (N+1)^6       *************************************************/
		 //start timer
		timer::start();
		array BruteForceSphericalGrid = constant(0, d_NoOfAnglesTheta, d_NoOfAnglesPhi, d_NoOfElements, PRECISION_COMPLEX);      // Angle Phi vs. Polar slices : Each Polar slice has No of angle Theta vs. Radial data
		ComputeDirectBruteForce3D(Image3D, d_NoOfElements, d_NoOfAnglesTheta, d_NoOfAnglesPhi, BruteForceSphericalGrid); // Filling up the solution data
		printf("Total time with brute force O((N+1)^6), elapsed seconds: %g\n", timer::stop());
		af_print(BruteForceSphericalGrid);

		int p__ = 1;																				  // Angle theta = 18, Phi = 18
		int q__ = 1;
		printf("\n Brute Force Solution 1st XX block\n");
		af_print(moddims(BruteForceSphericalGrid(p__, q__, span), 1, d_NoOfElements));										   // Line 1
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(p__, d_NoOfAnglesPhi - q__, span), 1, d_NoOfElements));					   // Line 2
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta - p__, q__, span), 1, d_NoOfElements));					   // Line 3
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta - p__, d_NoOfAnglesPhi - q__, span), 1, d_NoOfElements));   // Line 4
		printf("\n");

		printf("\n Brute Force Solution 1st XZ block\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 - p__, q__, span), 1, d_NoOfElements));					    // Line 1
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 - p__, d_NoOfAnglesPhi - q__, span), 1, d_NoOfElements));	// Line 2
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 + p__, q__, span), 1, d_NoOfElements));						// Line 3
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 + p__, d_NoOfAnglesPhi - q__, span), 1, d_NoOfElements));    // Line 4
		printf("\n");

		
		printf("\n Brute Force Solution 1st YY block\n");
		af_print(moddims(BruteForceSphericalGrid(p__, d_NoOfAnglesPhi / 2 - q__, span), 1, d_NoOfElements));					   // Line 1
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(p__, d_NoOfAnglesPhi / 2 + q__, span), 1, d_NoOfElements));					   // Line 2
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta - p__, d_NoOfAnglesPhi / 2 - q__, span), 1, d_NoOfElements));   // Line 3
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta - p__, d_NoOfAnglesPhi / 2 + q__, span), 1, d_NoOfElements));   // Line 4
		printf("\n");
		

		printf("\n Brute Force Solution 1st YZ block\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 - p__, d_NoOfAnglesPhi / 2 - q__, span), 1, d_NoOfElements));	// Line 1
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 - p__, d_NoOfAnglesPhi / 2 + q__, span), 1, d_NoOfElements));	// Line 2
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 + p__, d_NoOfAnglesPhi / 2 - q__, span), 1, d_NoOfElements));     // Line 3
		printf("\n");
		af_print(moddims(BruteForceSphericalGrid(d_NoOfAnglesTheta / 2 + p__, d_NoOfAnglesPhi / 2 + q__, span), 1, d_NoOfElements));	 // Line 4
		printf("\n");

		/******************************Begin Actual Spherical DFT****************************/
		double time_s = timeit(bench); // seconds
		printf("Timing with microsecond accurate clock (GPU) %f seconds\n", time_s);
		 
		timer::start();
		int d_NoOfLevelsTheta = ceil(float(d_NoOfAnglesTheta - 2) / 4);          // This is where partitioning begins
		int d_NoOfLevelsPhi = ceil(float(d_NoOfAnglesPhi - 2) / 4);

		array FinalSphericalGrid = constant(0, d_NoOfAnglesTheta, d_NoOfAnglesPhi, d_NoOfElements, PRECISION_COMPLEX);      // Angle Phi vs. Polar slices : Each Polar slice has No of elevation angle Theta vs. Radial data
		ComputeFastSphericalPolarFourierTransform(Image3D, FinalSphericalGrid, d_NoOfElements, d_NoOfAnglesTheta, d_NoOfLevelsTheta, d_NoOfAnglesPhi, d_NoOfLevelsPhi);
		printf("Total time for my Special solution O( 3 x K/4 x M/4 x (N+1)^3/2 x log2(N+1) ), elapsed seconds: %g\n", timer::stop());
		
		
		printf("\nBrute Force Special Grid\n");
		af_print(BruteForceSphericalGrid);
		printf("\nMy Special Solution\n");
		af_print(FinalSphericalGrid);

		
		af_print(FinalSphericalGrid(span, 0, span) - BruteForceSphericalGrid(span, 0, span));            // This is YZ slice anglePhi == 0
		
		af_print(FinalSphericalGrid(span, d_NoOfAnglesPhi / 2, span) - BruteForceSphericalGrid(span, d_NoOfAnglesPhi / 2, span));            // This is YZ slice anglePhi == 90

		af_print(FinalSphericalGrid(0, span, span) - BruteForceSphericalGrid(0, span, span));                  // This is the XY slice when angleTheta == 0 

		af_print(FinalSphericalGrid(d_NoOfAnglesTheta / 2, span, span) - BruteForceSphericalGrid(d_NoOfAnglesTheta / 2, span, span));
		

		printf("\nError between the direct brute force computation and the fast exact solution \n");
		af_print(BruteForceSphericalGrid - FinalSphericalGrid); 
		af_print(abs(sum(sum(sum(BruteForceSphericalGrid - FinalSphericalGrid)))));

	}
	catch (af::exception& e) {
		fprintf(stderr, "%s\n", e.what());
		throw;
	}

//#ifdef WIN32 // pause in Windows
//	if (!(argc == 2 && argv[1][0] == '-')) {
//		printf("hit [enter]...");
//		getchar();
//	}
//#endif
	return 0;
	
	
}





