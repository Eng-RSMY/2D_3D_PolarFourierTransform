#include "hip/hip_runtime.h"
/*
* Example of how to use the mxGPUArray API in a MEX file.  This example shows
* how to write a MEX function that takes a gpuArray as input and returns a
* gpuArray output for 2D Radon solution, e.g. B=mexFunction(A).
*
* by Syed Alam Abbas, 5/25/2015
*/
#include <arrayfire.h>
#include <af/util.h>
#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "math.h"

using namespace af;

//static const cdouble i_cdouble = { 0, 1 };
//static const  array i = constant(i_cdouble, 1, 1, c64);/* imaginary unit */
static  int isFirstRun_Uniform = 1;        // Flag to check if this is a first run on function  Compute2DColumnwise_FrFTUniform
static  int isFirstRun_Variable = 1;        // Flag to check if this is a first run on function  Compute2DComplementaryLines_FrFTVariableScales 
static  array PreMultiplicationFactor, PostMultiplicationFactor, Seq_En, BetaFactor;

enum SUPPORTED_PLATFORMS
{
	CUDA, OPENCL, CPU
};

/* Taking exp of complex numbers*/
array cexp(const array &in)
{
	if (!in.iscomplex()) return exp(in);
	return exp(real(in))*complex(cos(imag(in)), sin(imag(in)));
}

/* Multiplication of 2 complex numbers require 4 real multiplications*/
void SplitMultiplyComplex(array& A_Complex, array&  B_Complex, array& realRealPart, array& realImagPart, array& imagRealPart, array& imagImagPart)
{
	// Consider multiplication of complex numbers A_Complex = (a+ib); B_Complex = (c+id)	
	//array A_Complex, B_Complex;
	//array realRealPart;             // ac
	//array realImagPart;				// ad
	//array imagRealPart;				// bc
	//array imagImagPart;				// bd
	realRealPart = real(A_Complex)*real(B_Complex);
	realImagPart = real(A_Complex)*imag(B_Complex);
	imagRealPart = imag(A_Complex)*real(B_Complex);
	imagImagPart = imag(A_Complex)*imag(B_Complex);
}

/* Uniform FrFT for each column in Image*/
array Compute2DColumnwise_FrFTUniform(array & Image2D, array& ColumnScales_1D, int& d_NoOfElements_, int& d_NoOfScales)
{
	/*-----------------------------------Preparing Padded & Tiled Imag2D --------------------------------------------*/
	array Zeros = constant(0, d_NoOfElements_, d_NoOfElements_, f64);            // Generates on the device
	array Zero_Padded_Image2D = join(0, Image2D, Zeros);
	array Image2D_Tiled = tile(Zero_Padded_Image2D, 1, 1, d_NoOfScales);
	int N = d_NoOfElements_ - 1;
	if (isFirstRun_Uniform == 1)
	{
		/*-------------------------------------------Creating Index Cubes and Sequences----------------------------------------------------*/
		array leftSideIndexes = array(seq(0, N, 1)).as(f64);
		array rightSideIndexesOnes = -1 * array(seq(1, d_NoOfElements_, 1)).as(f64);
		array rightSideIndexesZeros = constant(0, d_NoOfElements_, 1, f64);
		array rightSideIndexesN_2 = constant(N / 2, d_NoOfElements_, 1, f64);

		array indexedElementsEn = join(0, leftSideIndexes, flip(rightSideIndexesOnes, 0));
		array indexedElementsPre = join(0, leftSideIndexes, rightSideIndexesN_2);     /* This is for Keeping pre and post multiplication factor upper half only*/
		array indexedElementsPost = join(0, leftSideIndexes, rightSideIndexesZeros);

		array indexedElements_Tiled_En = tile(pow(indexedElementsEn, 2), 1, d_NoOfElements_, d_NoOfScales);
		array indexedElements_Tiled_PreMulti = tile(indexedElementsPre - N / 2, 1, d_NoOfElements_, d_NoOfScales);
		array indexedElements_Tiled_PostMulti = tile(indexedElementsPost, 1, d_NoOfElements_, d_NoOfScales);

		/*--------------------------Creating FrFT scale cubes------------------------------------*/
		array ColumnScales_1D_Mods = moddims(ColumnScales_1D, 1, 1, d_NoOfScales);
		array ColumnScales_1D_Tiled_depth = tile(ColumnScales_1D_Mods, 2 * d_NoOfElements_, d_NoOfElements_, 1);
		array ColumnScales_1D_FullTiled = moddims(ColumnScales_1D_Tiled_depth, 2 * d_NoOfElements_, d_NoOfElements_, d_NoOfScales);

		/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
		//array imaginaryUnit_Tiled = tile(i, 2 * d_NoOfElements_, d_NoOfElements_, d_NoOfScales);
		Seq_En = cexp( complex(0, - af::Pi * indexedElements_Tiled_En * ColumnScales_1D_FullTiled / d_NoOfElements_));   /* E(n) as defined in the paper*/
		array Ones = constant(1, d_NoOfElements_, d_NoOfElements_, f64);
		array subtractValues = tile(join(0, Zeros, Ones), 1, 1, d_NoOfScales);			/* This is for  Keeping pre and post multiplication factor upper half only*/
		PreMultiplicationFactor = cexp( complex (0, af::Pi * indexedElements_Tiled_PreMulti * ColumnScales_1D_FullTiled * N / d_NoOfElements_)) - subtractValues;
		PostMultiplicationFactor = cexp( complex ( 0,  af::Pi *  indexedElements_Tiled_PostMulti * ColumnScales_1D_FullTiled * N / d_NoOfElements_)) - subtractValues;
		isFirstRun_Uniform = 0;
		//af::deviceGC();
	}
	/*--------------------Preprocessing Cubes-----------------------*/
	array Image2D_Tiled_PreMulti = Image2D_Tiled * PreMultiplicationFactor;
	array Image2D_Tiled_PreMulti_SeqEn = Image2D_Tiled_PreMulti * Seq_En;


	/*-------------------Computing Convolution--------------------*/
	array firstFFT_X = fft(Image2D_Tiled_PreMulti_SeqEn);
	array secondFFT_X = fft(conjg(Seq_En));
	array interim_FrFT_X = ifft(firstFFT_X * secondFFT_X);


	/*-------------------Postprocessing-----------------------------*/
	array  FrFT_Image_X = interim_FrFT_X * Seq_En * PostMultiplicationFactor;

	/*--------------------Grab only the top half elements drop overlapping------------------*/
	array FrFT_Image_X_Cube = FrFT_Image_X.rows(0, N);
	return FrFT_Image_X_Cube;
}

/* Variable FrFT for each column in Image*/
void Compute2DComplementaryLines_FrFTVariableScales(array & OneD_FrFT, array& ColumnScales_1D, array& final2DFrFTImage, array& final2DFrFTConjImage, int d_NoOfElements, int d_NoOfScales)
{
	int N = d_NoOfElements - 1;
	if (isFirstRun_Variable == 1)
	{
		array lineSpacing = array(seq(-N / 2, N / 2)).as(f64);
		array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
		array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
		array lineSpacing_Square_TiledLevel = tile(lineSpacing_Square, 1, 1, d_NoOfScales);
	
		//af_print(beta_Levels);
		array beta_Mods = moddims(ColumnScales_1D, 1, 1, d_NoOfScales);
		array beta_Tiled_depth = tile(beta_Mods, d_NoOfElements, d_NoOfElements, 1);
		//af_print(beta_Tiled_depth);
		array beta_Tiled = moddims(beta_Tiled_depth, d_NoOfElements, d_NoOfElements, d_NoOfScales);
	
		/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
		BetaFactor = cexp( complex(0, -2 * af::Pi * lineSpacing_Square_TiledLevel * beta_Tiled / d_NoOfElements));
	
		isFirstRun_Variable = 0;
		af::deviceGC();
	}
	// Consider multiplication of complex numbers A = (a+ib); B = (c+id)	
	array realRealPart;             // ac
	array realImagPart;				// ad 
	array imagRealPart;				// bc
	array imagImagPart;				// bd

	SplitMultiplyComplex(OneD_FrFT, BetaFactor, realRealPart, realImagPart, imagRealPart, imagImagPart);

	array tempSeq_X = sum(complex(realRealPart - imagImagPart, realImagPart + imagRealPart)); // sum(flip(FrFT_Image_X_Cube, 0) *BetaFactor);
	array tempSeqConj_X = sum(complex(realRealPart + imagImagPart, imagRealPart - realImagPart));//  sum(flip(FrFT_Image_X_Cube, 0) * conjg(BetaFactor));
	
	final2DFrFTImage = moddims(tempSeq_X, d_NoOfElements, d_NoOfScales).T();
	final2DFrFTConjImage = moddims(tempSeqConj_X, d_NoOfElements, d_NoOfScales).T();
}

/*
* High level Host code
* Computes the FrFT centered using the definition given in the paper,
"An exact and fast computation of Discrete Fourier Transform for polar grid and spherical grid"
by Syed Alam Abbas, 5/25/2015
*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	try {
		/* Initialize the MathWorks GPU API. */ 
		mxInitGPU();

		mexPrintf("Executing custom mex for computing 2D DFT on a Polar Grid using ArrayFire GPU accelerated library latest!");

		// Validate the input
		if (nrhs < 5 || nlhs < 2) {
			mexErrMsgTxt("Expected 5 inputs and 2 output.");
		}

		/*Input Variables*/
		const double* d_Image;
		int d_NoOfAngles;
		int d_NoOfLevels;
		int d_NoOfElements;
		mxGPUArray const * MxInputImage;
		mxArray* MxInputImageCPU;
		
		int PLATFORM = (size_t)mxGetScalar(prhs[4]);        // Given as an input PLATFORM

		switch (PLATFORM)                   // The settings change for input
		{
		case CUDA:
		case OPENCL:
			MxInputImage = mxGPUCreateFromMxArray(prhs[0]);   // GPU
			/* extract a pointer to the input data which is a real image on the device.*/
			d_Image = (double const *)(mxGPUGetDataReadOnly(MxInputImage));  // GPU 
			break;
		case CPU:
			MxInputImageCPU = mxDuplicateArray(prhs[0]);
			d_Image = mxGetPr(MxInputImageCPU);
			break;
		default:
			break;
		}


		/* Collect the input data from MATLAB MexArray RHS */
		d_NoOfAngles = (size_t)mxGetScalar(prhs[1]);            /*Check it, this should always be even*/
		d_NoOfLevels = (size_t)mxGetScalar(prhs[2]);
		d_NoOfElements = (size_t)mxGetScalar(prhs[3]);			/*Check it, this should always be odd*/
		int N = d_NoOfElements - 1;								/* it is always even as described in the paper*/

		/*********************Creating Array Fire object************************************/
		array Image(d_NoOfElements, d_NoOfElements, d_Image);


		/*--------------------------Creating Alpha levels------------------------------------*/
		array alpha_Levels = cos(af::Pi / ((double) d_NoOfAngles) * array(seq(1, d_NoOfLevels)).as(f64));

		/*--------------------------Creating Beta levels---------------------------------------*/
		array beta_Levels = sin(Pi / ((double)d_NoOfAngles) * array(seq(1, d_NoOfLevels)).as(f64));

		/*-------------------Precomputing the Essential Sequence Cubes :: All complex values here --------------------*/
		array lineSpacing = array(seq(-N / 2, N / 2)).as(f64);
		array lineSpacing_tiled_Y = tile(lineSpacing, 1, d_NoOfElements);
		array lineSpacing_Square = lineSpacing_tiled_Y * lineSpacing_tiled_Y.T();
		array ZeroNinty_Factor = cexp( complex( 0, -2 * af::Pi * lineSpacing_Square * 1 / d_NoOfElements));

		/*-------------------- First dimension uniform FrFT for each Image per level-----------------------*/
		array FrFT_Image_X_Cube = Compute2DColumnwise_FrFTUniform(Image.T(), alpha_Levels, d_NoOfElements, d_NoOfLevels);
		switch (PLATFORM)
		{
		case CUDA:
			af::deviceGC();
		default:
			break; 
		}

		array FrFT_Image_Y_Cube = Compute2DColumnwise_FrFTUniform((Image), alpha_Levels, d_NoOfElements, d_NoOfLevels);

		FrFT_Image_X_Cube = FrFT_Image_X_Cube.T();       // Now it needs operation to the other dimension
		FrFT_Image_Y_Cube = FrFT_Image_Y_Cube.T();

		
		/*--------------------Finally all computations for  the  Polar Grid-----------*/
		//   Computing for all the grid expect two special indexes
		array levelSeq = array(seq(0, d_NoOfLevels - 1)).as(f64);
		array finalIndexSeq1_X = 1 + levelSeq;
		array finalIndexSeq2_X = d_NoOfAngles - finalIndexSeq1_X;
		array finalIndexSeq3_Y = d_NoOfAngles / 2 - finalIndexSeq1_X;
		array finalIndexSeq4_Y = d_NoOfAngles / 2 + finalIndexSeq1_X;


		array finalSeq_X, finalSeqConj_X;
		Compute2DComplementaryLines_FrFTVariableScales((FrFT_Image_X_Cube), beta_Levels, finalSeq_X, finalSeqConj_X, d_NoOfElements, d_NoOfLevels);
		finalSeqConj_X = flip(finalSeqConj_X, 1);             // Special operation
		array finalSeq_Y, finalSeqConj_Y;
		Compute2DComplementaryLines_FrFTVariableScales(FrFT_Image_Y_Cube, beta_Levels, finalSeq_Y, finalSeqConj_Y, d_NoOfElements, d_NoOfLevels);

		// Removing just 2 redundant computations for 45 degree case
		if (0 == remainder(d_NoOfAngles, 4))
		{
			finalIndexSeq3_Y = finalIndexSeq3_Y.rows(0, d_NoOfLevels - 2);          // Removing just the last rows from 4 structures
			finalSeq_Y = finalSeq_Y.rows(0, d_NoOfLevels - 2);
			finalIndexSeq4_Y = finalIndexSeq4_Y.rows(0, d_NoOfLevels - 2);
			finalSeqConj_Y = finalSeqConj_Y.rows(0, d_NoOfLevels - 2);
		}

		//   Computing seperately for two special indexes
		double zeroIndex = 0;
		double nintyIndex = d_NoOfAngles / 2;
		double values[] = { zeroIndex, nintyIndex };
		array SpecialTwoIndexes(2, 1, values);

		array ZeroLineFrFT_Image_X_Cube = FrFT_Image_Y_Cube.slice(zeroIndex).col(N / 2);
		array NintyLineFrFT_Image_Y_Cube = FrFT_Image_X_Cube.slice(zeroIndex).col(N / 2);

		array DFTZeroLine = sum(tile(ZeroLineFrFT_Image_X_Cube, 1, d_NoOfElements) *ZeroNinty_Factor);
		array DFTNinetyLine = sum(tile((NintyLineFrFT_Image_Y_Cube), 1, d_NoOfElements) *ZeroNinty_Factor);
		array SpecialTwoLines = join(0, DFTZeroLine, DFTNinetyLine);

		array UnsortedIndexes = join(0, join(0, join(0, join(0, finalIndexSeq1_X, finalIndexSeq2_X), finalIndexSeq3_Y), finalIndexSeq4_Y), SpecialTwoIndexes);
		array tiledUnsortedIndexes = tile(UnsortedIndexes, 1, d_NoOfElements);
		array UnsortedPolarGrid = join(0, join(0, join(0, join(0, finalSeq_X, finalSeqConj_X), finalSeq_Y), finalSeqConj_Y), SpecialTwoLines);


		array FinalPolarGridReal;// = constant(0, d_NoOfElements, d_NoOfAngles, c64);
		array Output_Keys_Sorted;
		sort(Output_Keys_Sorted, FinalPolarGridReal, tiledUnsortedIndexes, real(UnsortedPolarGrid));

		array FinalPolarGridImag;// = constant(0, d_NoOfElements, d_NoOfAngles, c64);
		array Output_Keys_Sorted2;
		sort(Output_Keys_Sorted2, FinalPolarGridImag, tiledUnsortedIndexes, imag(UnsortedPolarGrid));


		mexPrintf("\nSuccessfully completed the computations of 2D DFT on a full Polar Grid  %d-by-%d!", d_NoOfAngles,d_NoOfElements);
		
		double* d_FinalPolarGridReal;       // Device pointer obtained from ArrayFire computations
		double* d_FinalPolarGridImag;		// Device pointer obtained from ArrayFire computations
		double* PolarGridReal_OUTPUT;       // MATLAB output pointer to be copied to the solution
		double* PolarGridImag_OUTPUT;		// MATLAB output pointer to be copied to the solution

		mwSize dims[] = { d_NoOfAngles, d_NoOfElements };

		switch (PLATFORM)                   // The settings change for input
		{
		case CUDA:
		case OPENCL:
			// Final processed double pointers
			d_FinalPolarGridReal = FinalPolarGridReal.device<double>();
			d_FinalPolarGridImag = FinalPolarGridImag.device<double>();

			/*output variables*/
			mxGPUArray* mxOutputRealPolarGridImage;
			mxGPUArray* mxOutputImagPolarGridImage;
			

			/* Create a GPUArray to hold the result and get its underlying pointer. */
			mxOutputRealPolarGridImage = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(MxInputImage),
				dims,
				mxGPUGetClassID(MxInputImage),
				mxGPUGetComplexity(MxInputImage),
				MX_GPU_DO_NOT_INITIALIZE);
			PolarGridReal_OUTPUT = (double *)(mxGPUGetData(mxOutputRealPolarGridImage));

			mxOutputImagPolarGridImage = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(MxInputImage),
				dims,
				mxGPUGetClassID(MxInputImage),
				mxGPUGetComplexity(MxInputImage),
				MX_GPU_DO_NOT_INITIALIZE);
			PolarGridImag_OUTPUT = (double *)(mxGPUGetData(mxOutputImagPolarGridImage));
			/* Copy processed Values from array object to MxArrayRealData */
			hipMemcpy(PolarGridReal_OUTPUT, d_FinalPolarGridReal, d_NoOfAngles*d_NoOfElements* sizeof(double), hipMemcpyDeviceToDevice);
			hipMemcpy(PolarGridImag_OUTPUT, d_FinalPolarGridImag, d_NoOfAngles*d_NoOfElements* sizeof(double), hipMemcpyDeviceToDevice);

			/* Wrap the result up as a MATLAB gpuArray for return. */
			plhs[0] = mxGPUCreateMxArrayOnGPU(mxOutputRealPolarGridImage);
			plhs[1] = mxGPUCreateMxArrayOnGPU(mxOutputImagPolarGridImage);
			/*
			* The mxGPUArray pointers are host-side structures that refer to device
			* data. These must be destroyed before leaving the MEX function.
			*/
			mxGPUDestroyGPUArray(MxInputImage);

			break;
		case CPU:
			// Final processed double pointers
			d_FinalPolarGridReal = FinalPolarGridReal.host<double>();   // Source
			d_FinalPolarGridImag = FinalPolarGridImag.host<double>();

			mxArray*  mxOutputRealPolarGridImageCPU;
			mxArray*  mxOutputImagPolarGridImageCPU;
			mxOutputRealPolarGridImageCPU = mxCreateNumericArray(2, dims, mxDOUBLE_CLASS, mxREAL);
			mxOutputImagPolarGridImageCPU = mxCreateNumericArray(2, dims, mxDOUBLE_CLASS, mxREAL);
			PolarGridReal_OUTPUT = mxGetPr(mxOutputRealPolarGridImageCPU);
			PolarGridImag_OUTPUT = mxGetPr(mxOutputImagPolarGridImageCPU);

			memcpy(PolarGridReal_OUTPUT, d_FinalPolarGridReal, d_NoOfAngles*d_NoOfElements* sizeof(double));
			memcpy(PolarGridImag_OUTPUT, d_FinalPolarGridImag, d_NoOfAngles*d_NoOfElements* sizeof(double));

			plhs[0] = mxOutputRealPolarGridImageCPU;
			plhs[1] = mxOutputImagPolarGridImageCPU;
			break;
		default:
			break;
		}


		mexPrintf("\nFinished processing custom CUDA mex with ArrayFire for computing 2D DFT on Polar Grid, Status = Success\n");

	}
	catch (af::exception &ex) {
		mexPrintf("%s\n", ex.what());
	}

	
}
 